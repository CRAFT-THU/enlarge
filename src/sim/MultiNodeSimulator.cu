#include "hip/hip_runtime.h"
/* This program is writen by qp09.
 * usually just for fun.
 * Sat October 24 2015
 */

#include <sys/time.h>
#include <stdio.h>
#include <assert.h>
#include <pthread.h>
#include <iostream>
#include <mpi.h>

#include "../utils/utils.h"
#include "../utils/TypeFunc.h"
#include "../gpu_utils/mem_op.h"
// #include "../gpu_utils/gpu_utils.h"
#include "../gpu_utils/runtime.h"
#include "../gpu_utils/GBuffers.h"
#include "../net/Network.h"
#include "../neuron/lif/LIFData.h"
#include "MultiNodeSimulator.h"
// #include "../gpu_utils/gpu_func.h"

using std::cout;
using std::endl;

MultiNodeSimulator::MultiNodeSimulator(Network *network, real dt) : Simulator(network, dt)
{
}

MultiNodeSimulator::~MultiNodeSimulator()
{
}

int run_node_gpu(DistriNetwork *network, CrossNodeData *cnd) {
	char log_filename[512];
	sprintf(log_filename, "sim_%d.gpu.log", network->_nodeIdx); 
	FILE *log_file = fopen(log_filename, "w+");
	assert(log_file != NULL);

	char v_filename[512];
	sprintf(v_filename, "v_%d.gpu.data", network->_nodeIdx); 
	FILE *v_file = fopen(v_filename, "w+");
	assert(v_file != NULL);

	checkCudaErrors(hipSetDevice(network->_nodeIdx));

	GNetwork *pNetCPU = network->_network;
	GNetwork *c_pNetGPU = copyGNetworkToGPU(pNetCPU);

	CrossNodeData * cnd_gpu = copyCNDtoGPU(cnd);

	int nTypeNum = c_pNetGPU->nTypeNum;
	int sTypeNum = c_pNetGPU->sTypeNum;
	int nodeNeuronNum = c_pNetGPU->pNeuronNums[nTypeNum];
	int allNeuronNum = pNetCPU->pConnection->nNum;
	int nodeSynapseNum = c_pNetGPU->pSynapseNums[sTypeNum];
	printf("Thread %d NeuronTypeNum: %d, SynapseTypeNum: %d\n", network->_nodeIdx, nTypeNum, sTypeNum);
	printf("Thread %d NeuronNum: %d, SynapseNum: %d\n", network->_nodeIdx, nodeNeuronNum, nodeSynapseNum);

	int maxDelay = pNetCPU->pConnection->maxDelay;
	int minDelay = pNetCPU->pConnection->minDelay;

	printf("Thread %d MaxDelay: %d MinDelay: %d\n", network->_nodeIdx, maxDelay,  minDelay);


	GBuffers *buffers = alloc_buffers(allNeuronNum, nodeSynapseNum, pNetCPU->pConnection->maxDelay, network->_dt);

	BlockSize *updateSize = getBlockSize(allNeuronNum, nodeSynapseNum);

#ifdef LOG_DATA
	real *c_vm = hostMalloc<real>(nodeNeuronNum);
	int life_idx = getIndex(c_pNetGPU->pNTypes, nTypeNum, LIF);
	int copy_idx = -1;
	real *c_g_vm = NULL;

	if (life_idx >= 0) {
		LIFData *c_g_lif = copyFromGPU<LIFData>(static_cast<LIFData *>(c_pNetGPU->ppNeurons[life_idx]), 1);
		c_g_vm = c_g_lif->pV_m;
		copy_idx = life_idx;
	} else {
	}
#endif

	for (int i=0; i<nTypeNum; i++) {
		cout << "Thread " << network->_nodeIdx << " " << c_pNetGPU->pNTypes[i] << ": <<<" << updateSize[c_pNetGPU->pNTypes[i]].gridSize << ", " << updateSize[c_pNetGPU->pNTypes[i]].blockSize << ">>>" << endl;
	}
	for (int i=0; i<sTypeNum; i++) {
		cout << "Thread " << network->_nodeIdx << " " << c_pNetGPU->pSTypes[i] << ": <<<" << updateSize[c_pNetGPU->pSTypes[i]].gridSize << ", " << updateSize[c_pNetGPU->pSTypes[i]].blockSize << ">>>" << endl;
	}

	int * c_g_idx2index = copyToGPU<int>(network->_crossnodeMap->_idx2index, allNeuronNum);
	int * c_g_cross_index2idx = copyToGPU<int>(network->_crossnodeMap->_crossnodeIndex2idx, network->_crossnodeMap->_crossSize);
	int * c_g_global_cross_data = gpuMalloc<int>(allNeuronNum * network->_nodeNum);
	int * c_g_fired_n_num = gpuMalloc<int>(network->_nodeNum);

	vector<int> firedInfo;
	struct timeval ts, te;
	gettimeofday(&ts, NULL);
	for (int time=0; time<network->_simCycle; time++) {
		update_time<<<1, 1>>>(c_pNetGPU->pConnection, time, buffers->c_gFiredTableSizes);

		for (int i=0; i<nTypeNum; i++) {
			assert(c_pNetGPU->pNeuronNums[i+1]-c_pNetGPU->pNeuronNums[i] > 0);
			cudaUpdateType[c_pNetGPU->pNTypes[i]](c_pNetGPU->pConnection, c_pNetGPU->ppNeurons[i], buffers->c_gNeuronInput, buffers->c_gNeuronInput_I, buffers->c_gFiredTable, buffers->c_gFiredTableSizes, c_pNetGPU->pNeuronNums[i+1]-c_pNetGPU->pNeuronNums[i], c_pNetGPU->pNeuronNums[i], time, &updateSize[c_pNetGPU->pNTypes[i]]);
		}

		hipMemset(cnd_gpu->_send_num, 0, sizeof(int)*(cnd_gpu->_node_num));
		cudaGenerateCND<<<(allNeuronNum+MAX_BLOCK_SIZE-1)/MAX_BLOCK_SIZE, MAX_BLOCK_SIZE>>>(c_pNetGP->pConnection, buffers->c_gFiredTable, buffers->c_gFiredTableSizes, c_g_idx2index, c_g_cross_index2idx, cnd_gpu->_send_data, cnd_gpu->_send_offset, cnd_gpu->_send_num, network->_nodeNum, time);

		// checkCudaErrors(hipMemcpy(gCrossDataGPU->_firedNum + network->_nodeIdx * network->_nodeNum, c_g_fired_n_num, sizeof(int)*network->_nodeNum, hipMemcpyDeviceToHost));

		checkCudaErrors(hipMemcpy(cnd->_send_num, cnd_gpu->_send_num, sizeof(int)*(cnd->_node_num), hipMemcpyDeviceToHost));
		for (int i=0; i< network->_nodeNum; i++) {

			if (cnd->_send_[i] > 0) {
			}
		}

#ifdef LOG_DATA
		int currentIdx = time%(maxDelay+1);

		int copySize = 0;
		copyFromGPU<int>(&copySize, buffers->c_gFiredTableSizes + currentIdx, 1);
		if (copySize > 0) {
			copyFromGPU<int>(buffers->c_neuronsFired, buffers->c_gFiredTable + (allNeuronNum*currentIdx), copySize);
		}

		if (copy_idx >= 0 && (c_pNetGPU->pNeuronNums[copy_idx+1]-c_pNetGPU->pNeuronNums[copy_idx]) > 0) {
			copyFromGPU<real>(c_vm, c_g_vm, c_pNetGPU->pNeuronNums[copy_idx+1]-c_pNetGPU->pNeuronNums[copy_idx]);
		}
#endif

		for (int i=0; i<sTypeNum; i++) {
			assert(c_pNetGPU->pSynapseNums[i+1]-c_pNetGPU->pSynapseNums[i] > 0);
			cudaUpdateType[c_pNetGPU->pSTypes[i]](c_pNetGPU->pConnection, c_pNetGPU->ppSynapses[i], buffers->c_gNeuronInput, buffers->c_gNeuronInput_I, buffers->c_gFiredTable, buffers->c_gFiredTableSizes, c_pNetGPU->pSynapseNums[i+1]-c_pNetGPU->pSynapseNums[i], c_pNetGPU->pSynapseNums[i], time, &updateSize[c_pNetGPU->pSTypes[i]]);
		}

		for (int i=0; i< network->_nodeNum; i++) {
			// int i2idx = network->_nodeIdx + network->_nodeNum * i;
			// if (gCrossDataGPU->_firedNum[i2idx] > 0) {
			// 	int num = gCrossDataGPU->_firedNum[i2idx];
			// 	cudaAddCrossNeurons<<<(num+MAX_BLOCK_SIZE-1)/MAX_BLOCK_SIZE, MAX_BLOCK_SIZE>>>(c_pNetGPU->pConnection, buffers->c_gFiredTable, buffers->c_gFiredTableSizes, gCrossDataGPU->_firedArrays[i2idx], gCrossDataGPU->_firedNum[i2idx], time);
			// }
		}
		
#ifdef LOG_DATA
		for (int i=0; i<copySize; i++) {
			fprintf(log_file, "%d ", buffers->c_neuronsFired[i]);
		}
		fprintf(log_file, "\n");

		for (int i=0; i<c_pNetGPU->pNeuronNums[copy_idx+1] - c_pNetGPU->pNeuronNums[copy_idx]; i++) {
			fprintf(v_file, "%.10lf \t", c_vm[i]);
		}
		fprintf(v_file, "\n");
#endif
	}
	gettimeofday(&te, NULL);
	long seconds = te.tv_sec - ts.tv_sec;
	long hours = seconds/3600;
	seconds = seconds%3600;
	long minutes = seconds/60;
	seconds = seconds%60;
	long uSeconds = te.tv_usec - ts.tv_usec;
	if (uSeconds < 0) {
		uSeconds += 1000000;
		seconds = seconds - 1;
	}

	printf("Thread %d Simulation finesed in %ld:%ld:%ld.%06lds\n", network->_nodeIdx, hours, minutes, seconds, uSeconds);

	int *rate = (int*)malloc(sizeof(int)*nodeNeuronNum);
	copyFromGPU<int>(rate, buffers->c_gFireCount, nodeNeuronNum);

	char fire_filename[512];
	sprintf(fire_filename, "fire_%d.gpu.count", network->_nodeIdx); 
	FILE *rate_file = fopen(fire_filename, "w+");
	if (rate_file == NULL) {
		printf("Open file Sim.log failed\n");
		return 0;
	}

	for (int i=0; i<nodeNeuronNum; i++) {
		fprintf(rate_file, "%d \t", rate[i]);
	}

	free(rate);
	fclose(rate_file);

	fclose(log_file);
	fclose(v_file);

	free_buffers(buffers);
	freeGNetworkGPU(c_pNetGPU);

	return 0;
}

int MultiNodeSimulator::run(real time, FireInfo &log)
{

	int sim_cycle = round(time/_dt);
	reset();

	MPI_Comm_rank(MPI_COMM_WORLD, &node_id);
	MPI_Comm_size(MPI_COMM_WORLD, &node_num);
	char processor_name[MPI_MAX_PROCESSOR_NAME];
	int name_len;
	MPI_Get_processor_name(processor_name, &name_len);
	printf("Processor %s, rank %d out of %d processors\n", processor_name, node_id, node_num);

	SimInfo info(_dt);

	DistriNetwork *network = NULL;
	CrossNodeData *data = NULL;

	if (node_id == 0) {
		_network->setNodeNum(node_num);
		DistriNetwork *node_nets = _network->buildNetworks(info);
		CrossNodeData *node_datas = _network->arrangeNodeData(node_num);

		for (int i=0; i<node_num; i++) {
			node_nets[i]._simCycle = sim_cycle;
			node_nets[i]._nodeIdx = i;
			node_nets[i]._nodeNum = node_num;
			node_nets[i]._dt = _dt;
		}

		network = &(node_nets[0]);
		data = &(node_datas[0]);
		allocDataCND(data);

		for (int i=1; i<node_num; i++) {
			sendDistriNet(&(node_nets[i]), i, DATA_TAG, MPI_COMM_WORLD);
			sendCND(&(node_datas[i]), i, DATA_TAG + DNET_TAG, MPI_COMM_WORLD);
		}
	} else {
		network = recvDistriNet(0, DATA_TAG, MPI_COMM_WORLD);
		data = recvCND(0, DATA_TAG + DNET_TAG, MPI_COMM_WORLD);
	}

	MPI_Barrier(MPI_COMM_WORLD);
	run_node_gpu(network, data);

	return 0;
}

