#include "hip/hip_runtime.h"
/* This program is writen by qp09.
 * usually just for fun.
 * Sat October 24 2015
 */

#include <sys/time.h>
#include <stdio.h>
#include <assert.h>
#include <pthread.h>
#include <iostream>

#include "../utils/utils.h"
#include "../utils/TypeFunc.h"
#include "../gpu_utils/mem_op.h"
#include "../gpu_utils/runtime.h"
#include "../gpu_utils/GBuffers.h"
#include "../net/Network.h"
#include "../neuron/lif/LIFData.h"

#include "MultiGPUSimulator.h"

using std::cout;
using std::endl;

pthread_barrier_t gpuCycleBarrier;

CrossNodeDataGPU * gCrossDataGPU;

MultiGPUSimulator::MultiGPUSimulator(Network *network, real dt) : Simulator(network, dt)
{
}

MultiGPUSimulator::~MultiGPUSimulator()
{
}

void *run_thread_gpu(void *para);

int MultiGPUSimulator::run(real time, FireInfo &log)
{
	int sim_cycle = round(time/_dt);
	reset();

	int device_count = 4;
	checkCudaErrors(hipGetDeviceCount(&device_count));
	assert(device_count != 0);
	for (int i=0; i<device_count; i++) {
		for (int j=0; j<device_count; j++) {
			if (i!=j) {
				int access = 0;
				checkCudaErrors(hipDeviceCanAccessPeer(&access, i, j));
				if (access == 1) {
					checkCudaErrors(hipSetDevice(i));
					checkCudaErrors(hipDeviceEnablePeerAccess(j, 0));
				}
			}
		}
	}
	checkCudaErrors(hipSetDevice(0));

	pthread_barrier_init(&gpuCycleBarrier, NULL, device_count);

	// MultiNetwork multiNet(_network, device_count);
	_network->setNodeNum(device_count);

	SimInfo info(_dt);
	DistriNetwork *node_nets = _network->buildNetworks(info);
	assert(node_nets != NULL);
	gCrossDataGPU = _network->arrangeCrossNodeDataGPU(device_count);
	assert(gCrossDataGPU != NULL);

	pthread_t *thread_ids = (pthread_t *)malloc(sizeof(pthread_t) * device_count);
	assert(thread_ids != NULL);


	for (int i=0; i<device_count; i++) {
		node_nets[i]._simCycle = sim_cycle;
		node_nets[i]._nodeIdx = i;
		node_nets[i]._nodeNum = device_count;
		node_nets[i]._dt = _dt;


		int ret = pthread_create(&(thread_ids[i]), NULL, &run_thread_gpu, (void*)&(node_nets[i]));
		assert(ret == 0);
	}

	for (int i=0; i<device_count; i++) {
		pthread_join(thread_ids[i], NULL);
	}

	pthread_barrier_destroy(&gpuCycleBarrier);

	return 0;
}

void * run_thread_gpu(void *para) {
	DistriNetwork *network = (DistriNetwork*)para;

	char log_filename[512];
	sprintf(log_filename, "sim_%d.gpu.log", network->_nodeIdx); 
	FILE *log_file = fopen(log_filename, "w+");
	assert(log_file != NULL);

	char v_filename[512];
	sprintf(v_filename, "v_%d.gpu.data", network->_nodeIdx); 
	FILE *v_file = fopen(v_filename, "w+");
	assert(v_file != NULL);

	checkCudaErrors(hipSetDevice(network->_nodeIdx));

	GNetwork *pNetCPU = network->_network;
	GNetwork *c_pNetGPU = copyNetworkToGPU(pNetCPU);

	int nTypeNum = c_pNetGPU->nTypeNum;
	int sTypeNum = c_pNetGPU->sTypeNum;
	int nodeNeuronNum = c_pNetGPU->pNeuronNums[nTypeNum];
	int allNeuronNum = pNetCPU->pConnection->nNum;
	int nodeSynapseNum = c_pNetGPU->pSynapseNums[sTypeNum];
	printf("Thread %d NeuronTypeNum: %d, SynapseTypeNum: %d\n", network->_nodeIdx, nTypeNum, sTypeNum);
	printf("Thread %d NeuronNum: %d, SynapseNum: %d\n", network->_nodeIdx, nodeNeuronNum, nodeSynapseNum);

	//int dataOffset = network->_nodeIdx * network->_nodeNum;
	//int dataIdx = network->_nodeIdx * network->_nodeNum + network->_nodeIdx;

	int maxDelay = pNetCPU->pConnection->maxDelay;
	int minDelay = pNetCPU->pConnection->minDelay;
	// int deltaDelay = maxDelay - minDelay;
	// int deltaDelay = pNetCPU->pConnection->maxDelay - pNetCPU->pConnection->minDelay;
	printf("Thread %d MaxDelay: %d MinDelay: %d\n", network->_nodeIdx, maxDelay,  minDelay);

	// init_connection<<<1, 1>>>(c_pNetGPU->pConnection);

	GBuffers *buffers = alloc_buffers(allNeuronNum, nodeSynapseNum, pNetCPU->pConnection->maxDelay, network->_dt);

	BlockSize *updateSize = getBlockSize(allNeuronNum, nodeSynapseNum);

#ifdef LOG_DATA
	real *c_vm = hostMalloc<real>(nodeNeuronNum);
	int life_idx = getIndex(c_pNetGPU->pNTypes, nTypeNum, LIF);
	int copy_idx = -1;
	real *c_g_vm = NULL;

	if (life_idx >= 0) {
		LIFData *c_g_lif = copyFromGPU<LIFData>(static_cast<LIFData *>(c_pNetGPU->ppNeurons[life_idx]), 1);
		c_g_vm = c_g_lif->pV_m;
		copy_idx = life_idx;
	} else {
	}
#endif

	for (int i=0; i<nTypeNum; i++) {
		cout << "Thread " << network->_nodeIdx << " " << c_pNetGPU->pNTypes[i] << ": <<<" << updateSize[c_pNetGPU->pNTypes[i]].gridSize << ", " << updateSize[c_pNetGPU->pNTypes[i]].blockSize << ">>>" << endl;
	}
	for (int i=0; i<sTypeNum; i++) {
		cout << "Thread " << network->_nodeIdx << " " << c_pNetGPU->pSTypes[i] << ": <<<" << updateSize[c_pNetGPU->pSTypes[i]].gridSize << ", " << updateSize[c_pNetGPU->pSTypes[i]].blockSize << ">>>" << endl;
	}

	//int * c_g_cross_id = gpuMalloc<int>(global_cross_data[dataIdx]._max_n_num); 

	int * c_g_idx2index = copyToGPU<int>(network->_crossnodeMap->_idx2index, allNeuronNum);
	int * c_g_cross_index2idx = copyToGPU<int>(network->_crossnodeMap->_crossnodeIndex2idx, network->_crossnodeMap->_crossSize);
	int * c_g_global_cross_data = gpuMalloc<int>(allNeuronNum * network->_nodeNum);
	int * c_g_fired_n_num = gpuMalloc<int>(network->_nodeNum);

	vector<int> firedInfo;
	struct timeval ts, te;
	//struct timeval t0, t1, t2, t3, t4, t5,/* t6,*/ t7, t8, t9;
	//double barrier1_time = 0, gpu_cpy_time = 0, peer_cpy_time = 0, barrier2_time=0, copy_time = 0;
	gettimeofday(&ts, NULL);
	for (int time=0; time<network->_simCycle; time++) {
		update_time<<<1, 1>>>(c_pNetGPU->pConnection, time, buffers->c_gFiredTableSizes);

		for (int i=0; i<nTypeNum; i++) {
			assert(c_pNetGPU->pNeuronNums[i+1]-c_pNetGPU->pNeuronNums[i] > 0);
			cudaUpdateType[c_pNetGPU->pNTypes[i]](c_pNetGPU->pConnection, c_pNetGPU->ppNeurons[i], buffers->c_gNeuronInput, buffers->c_gNeuronInput_I, buffers->c_gFiredTable, buffers->c_gFiredTableSizes, c_pNetGPU->pNeuronNums[i+1]-c_pNetGPU->pNeuronNums[i], c_pNetGPU->pNeuronNums[i], time, &updateSize[c_pNetGPU->pNTypes[i]]);
		}

		//gettimeofday(&t0, NULL);
		pthread_barrier_wait(&gpuCycleBarrier);
		//gettimeofday(&t1, NULL);
		//barrier1_time += (t1.tv_sec - t0.tv_sec) + (t1.tv_usec - t0.tv_usec)/1000000.0;
		hipMemset(c_g_fired_n_num, 0, sizeof(int)*network->_nodeNum);
		//hipDeviceSynchronize();
		//gettimeofday(&t2, NULL);
		//cudaDeliverNeurons(c_g_idx2index, c_g_cross_index2idx, c_g_global_cross_data, c_g_fired_n_num, network->_nodeNum, allNeuronNum);
		//for (int i=0; i<network->_nodeNum; i++) {
		//	int offset = i * network->_nodeNum + network->_nodeIdx; 
		//	copyFromGPU<int>(&(global_cross_data[offset]._fired_n_num), c_g_fired_n_num + i, 1);
		//	if (global_cross_data[offset]._fired_n_num > 0) {
		//		copyFromGPU<int>(global_cross_data[offset]._fired_n_idxs, c_g_global_cross_data + allNeuronNum * i, global_cross_data[offset]._fired_n_num);
		//	}
		//}

		cudaDeliverNeurons<<<(allNeuronNum+MAX_BLOCK_SIZE-1)/MAX_BLOCK_SIZE, MAX_BLOCK_SIZE>>>(c_pNetGPU->pConnection, buffers->c_gFiredTable, buffers->c_gFiredTableSizes, c_g_idx2index, c_g_cross_index2idx, c_g_global_cross_data, c_g_fired_n_num, network->_nodeNum, time);

		checkCudaErrors(hipMemcpy(gCrossDataGPU->_firedNum + network->_nodeIdx * network->_nodeNum, c_g_fired_n_num, sizeof(int)*network->_nodeNum, hipMemcpyDeviceToHost));
		//gettimeofday(&t3, NULL);

		for (int i=0; i< network->_nodeNum; i++) {
			int idx2i = network->_nodeIdx * network->_nodeNum + i;
			assert(gCrossDataGPU->_firedNum[idx2i] <= gCrossDataGPU->_maxNum[idx2i]);
			if (gCrossDataGPU->_firedNum[idx2i] > 0) {
				checkCudaErrors(hipMemcpyPeer(gCrossDataGPU->_firedArrays[idx2i], i, c_g_global_cross_data + allNeuronNum * i, network->_nodeIdx, gCrossDataGPU->_firedNum[idx2i] * sizeof(int)));
			}
		}
		//gettimeofday(&t7, NULL);

		//gpu_cpy_time += (t3.tv_sec - t2.tv_sec) + (t3.tv_usec - t2.tv_usec)/1000000.0;
		//peer_cpy_time += (t7.tv_sec - t3.tv_sec) + (t7.tv_usec - t3.tv_usec)/1000000.0;

#ifdef LOG_DATA
		int currentIdx = time%(maxDelay+1);

		int copySize = 0;
		copyFromGPU<int>(&copySize, buffers->c_gFiredTableSizes + currentIdx, 1);
		if (copySize > 0) {
			copyFromGPU<int>(buffers->c_neuronsFired, buffers->c_gFiredTable + (allNeuronNum*currentIdx), copySize);
		}

		if (copy_idx >= 0 && (c_pNetGPU->pNeuronNums[copy_idx+1]-c_pNetGPU->pNeuronNums[copy_idx]) > 0) {
			copyFromGPU<real>(c_vm, c_g_vm, c_pNetGPU->pNeuronNums[copy_idx+1]-c_pNetGPU->pNeuronNums[copy_idx]);
		}
#endif

		for (int i=0; i<sTypeNum; i++) {
			assert(c_pNetGPU->pSynapseNums[i+1]-c_pNetGPU->pSynapseNums[i] > 0);
			cudaUpdateType[c_pNetGPU->pSTypes[i]](c_pNetGPU->pConnection, c_pNetGPU->ppSynapses[i], buffers->c_gNeuronInput, buffers->c_gNeuronInput_I, buffers->c_gFiredTable, buffers->c_gFiredTableSizes, c_pNetGPU->pSynapseNums[i+1]-c_pNetGPU->pSynapseNums[i], c_pNetGPU->pSynapseNums[i], time, &updateSize[c_pNetGPU->pSTypes[i]]);
		}
		//hipDeviceSynchronize();

		//gettimeofday(&t4, NULL);
		pthread_barrier_wait(&gpuCycleBarrier);
		//gettimeofday(&t5, NULL);
		//barrier2_time += (t5.tv_sec - t4.tv_sec) + (t5.tv_usec - t4.tv_usec)/1000000.0;

		//gettimeofday(&t6, NULL);
		//collectNeurons();
		//gettimeofday(&t7, NULL);
		//cpu_cpy_time += (t7.tv_sec - t6.tv_sec) + (t7.tv_usec - t6.tv_usec)/1000000.0;
		
		//gettimeofday(&t8, NULL);
		//if (global_cross_data[dataIdx]._fired_n_num > 0) {
		//	copyToGPU(c_g_cross_id, global_cross_data[dataIdx]._fired_n_idxs, global_cross_data[dataIdx]._fired_n_num);
		//	addCrossNeurons(c_g_cross_id, global_cross_data[dataIdx]._fired_n_num);
		//}
		for (int i=0; i< network->_nodeNum; i++) {
			int i2idx = network->_nodeIdx + network->_nodeNum * i;
			if (gCrossDataGPU->_firedNum[i2idx] > 0) {
				int num = gCrossDataGPU->_firedNum[i2idx];
				cudaAddCrossNeurons<<<(num+MAX_BLOCK_SIZE-1)/MAX_BLOCK_SIZE, MAX_BLOCK_SIZE>>>(c_pNetGPU->pConnection, buffers->c_gFiredTable, buffers->c_gFiredTableSizes, gCrossDataGPU->_firedArrays[i2idx], gCrossDataGPU->_firedNum[i2idx], time);
			}
		}
		
		//gettimeofday(&t9, NULL);
		//copy_time += (t9.tv_sec - t8.tv_sec) + (t9.tv_usec - t8.tv_usec)/1000000.0;

#ifdef LOG_DATA
		for (int i=0; i<copySize; i++) {
			fprintf(log_file, "%d ", buffers->c_neuronsFired[i]);
		}
		fprintf(log_file, "\n");

		for (int i=0; i<c_pNetGPU->pNeuronNums[copy_idx+1] - c_pNetGPU->pNeuronNums[copy_idx]; i++) {
			fprintf(v_file, "%.10lf \t", c_vm[i]);
		}
		fprintf(v_file, "\n");
#endif

		//pthread_barrier_wait(&gpuCycleBarrier);
	}
	pthread_barrier_wait(&gpuCycleBarrier);
	gettimeofday(&te, NULL);
	long seconds = te.tv_sec - ts.tv_sec;
	long hours = seconds/3600;
	seconds = seconds%3600;
	long minutes = seconds/60;
	seconds = seconds%60;
	long uSeconds = te.tv_usec - ts.tv_usec;
	if (uSeconds < 0) {
		uSeconds += 1000000;
		seconds = seconds - 1;
	}

	printf("Thread %d Simulation finesed in %ld:%ld:%ld.%06lds\n", network->_nodeIdx, hours, minutes, seconds, uSeconds);
	//printf("Thread %d cost : barrier1 %lf, DtoH %lf, DtoD %lf, barrier2 %lf, HtoD %lf\n", network->_nodeIdx, barrier1_time, gpu_cpy_time, peer_cpy_time, barrier2_time, copy_time);

	int *rate = (int*)malloc(sizeof(int)*nodeNeuronNum);
	copyFromGPU<int>(rate, buffers->c_gFireCount, nodeNeuronNum);

	char fire_filename[512];
	sprintf(fire_filename, "fire_%d.gpu.count", network->_nodeIdx); 
	FILE *rate_file = fopen(fire_filename, "w+");
	if (rate_file == NULL) {
		printf("Open file Sim.log failed\n");
		return NULL;
	}

	for (int i=0; i<nodeNeuronNum; i++) {
		fprintf(rate_file, "%d \t", rate[i]);
	}

	free(rate);
	fclose(rate_file);

	fclose(log_file);
	fclose(v_file);

	free_buffers(buffers);
	freeNetworkGPU(c_pNetGPU);

	return NULL;
}


int MultiGPUSimulator::run_single(real time, FireInfo &log)
{
	int sim_cycle = round(time/_dt);
	reset();

	int device_count = 4;

	checkCudaErrors(hipSetDevice(0));

	_network->setNodeNum(device_count);

	SimInfo info(_dt);
	DistriNetwork *node_nets = _network->buildNetworks(info);
	assert(node_nets != NULL);
	gCrossDataGPU = _network->arrangeCrossNodeDataGPU(device_count);
	assert(gCrossDataGPU != NULL);


	for (int d=0; d<device_count; d++) {
		node_nets[d]._simCycle = sim_cycle;
		node_nets[d]._nodeIdx = d;
		node_nets[d]._nodeNum = device_count;
		node_nets[d]._dt = _dt;
	}


	FILE **log_file = (FILE **)malloc(sizeof(FILE*)*device_count);
	FILE **v_file = (FILE **)malloc(sizeof(FILE*)*device_count);

	GNetwork **c_pNetGPU =  (GNetwork **)malloc(sizeof(GNetwork*)*device_count);

	GBuffers **buffers = (GBuffers **)malloc(sizeof(GBuffers*)*device_count);

	BlockSize **updateSize = (BlockSize **)malloc(sizeof(BlockSize*)*device_count);
	int **c_g_idx2index = (int **)malloc(sizeof(int*)*device_count);
	int **c_g_cross_index2idx = (int **)malloc(sizeof(int*)*device_count);
	int **c_g_global_cross_data = (int **)malloc(sizeof(int*)*device_count);
	int **c_g_fired_n_num = (int **)malloc(sizeof(int*)*device_count);

	real **c_vm = (real **)malloc(sizeof(real*)*device_count);
	real **c_g_vm = (real **)malloc(sizeof(real*)*device_count);
	int *copy_idx = (int *)malloc(sizeof(int)*device_count);

	char log_filename[512];
	char v_filename[512];
	for (int d=0; d<device_count; d++) {
		sprintf(log_filename, "sim_%d.log", node_nets[d]._nodeIdx); 
		log_file[d] = fopen(log_filename, "w+");
		assert(log_file[d] != NULL);

		sprintf(v_filename, "v_%d.data", node_nets[d]._nodeIdx); 
		v_file[d] = fopen(v_filename, "w+");
		assert(v_file != NULL);

		GNetwork *pNetCPU = node_nets[d]._network;
		c_pNetGPU[d] = copyNetworkToGPU(pNetCPU);

		int nTypeNum = c_pNetGPU[d]->nTypeNum;
		int sTypeNum = c_pNetGPU[d]->sTypeNum;
		int nodeNeuronNum = c_pNetGPU[d]->pNeuronNums[nTypeNum];
		int allNeuronNum = c_pNetGPU[d]->pConnection->nNum;
		int nodeSynapseNum = c_pNetGPU[d]->pSynapseNums[sTypeNum];
		printf("Thread %d NeuronTypeNum: %d, SynapseTypeNum: %d\n", node_nets[d]._nodeIdx, nTypeNum, sTypeNum);
		printf("Thread %d NeuronNum: %d, SynapseNum: %d\n", node_nets[d]._nodeIdx, nodeNeuronNum, nodeSynapseNum);

		int maxDelay = pNetCPU->pConnection->maxDelay;
		int minDelay = pNetCPU->pConnection->minDelay;
		printf("Thread %d MaxDelay: %d MinDelay: %d\n", node_nets[d]._nodeIdx, maxDelay,  minDelay);

		buffers[d] = alloc_buffers(allNeuronNum, nodeSynapseNum, pNetCPU->pConnection->maxDelay, node_nets[d]._dt);

		updateSize[d] = getBlockSize(allNeuronNum, nodeSynapseNum);

#ifdef LOG_DATA
		c_vm[d] = hostMalloc<real>(nodeNeuronNum);
		int life_idx = getIndex(c_pNetGPU[d]->pNTypes, nTypeNum, LIF);
		copy_idx[d] = -1;

		if (life_idx >= 0) {
			LIFData *c_g_lif = copyFromGPU<LIFData>(static_cast<LIFData *>(c_pNetGPU[d]->ppNeurons[life_idx]), 1);
			c_g_vm[d] = c_g_lif->pV_m;
			copy_idx[d] = life_idx;
		} else {
		}
#endif

		for (int i=0; i<nTypeNum; i++) {
			cout << "Thread " << node_nets[d]._nodeIdx << " " << c_pNetGPU[d]->pNTypes[i] << ": <<<" << updateSize[d][c_pNetGPU[d]->pNTypes[i]].gridSize << ", " << updateSize[d][c_pNetGPU[d]->pNTypes[i]].blockSize << ">>>" << endl;
		}
		for (int i=0; i<sTypeNum; i++) {
			cout << "Thread " << node_nets[d]._nodeIdx << " " << c_pNetGPU[d]->pSTypes[i] << ": <<<" << updateSize[d][c_pNetGPU[d]->pSTypes[i]].gridSize << ", " << updateSize[d][c_pNetGPU[d]->pSTypes[i]].blockSize << ">>>" << endl;
		}

		c_g_idx2index[d] = copyToGPU<int>(node_nets[d]._crossnodeMap->_idx2index, allNeuronNum);
		c_g_cross_index2idx[d] = copyToGPU<int>(node_nets[d]._crossnodeMap->_crossnodeIndex2idx, node_nets[d]._crossnodeMap->_crossSize);
		c_g_global_cross_data[d] = gpuMalloc<int>(allNeuronNum * node_nets[d]._nodeNum);
		c_g_fired_n_num[d] = gpuMalloc<int>(node_nets[d]._nodeNum);
	}


	vector<int> firedInfo;
	struct timeval ts, te;
	gettimeofday(&ts, NULL);
	for (int time=0; time<node_nets[0]._simCycle; time++) {
		for (int d=0; d<device_count; d++) {
			update_time<<<1, 1>>>(c_pNetGPU[d]->pConnection, time, buffers[d]->c_gFiredTableSizes);

			for (int i=0; i<c_pNetGPU[d]->nTypeNum; i++) {
				assert(c_pNetGPU[d]->pNeuronNums[i+1]-c_pNetGPU[d]->pNeuronNums[i] > 0);
				cudaUpdateType[c_pNetGPU[d]->pNTypes[i]](c_pNetGPU[d]->pConnection, c_pNetGPU[d]->ppNeurons[i], buffers[d]->c_gNeuronInput, buffers[d]->c_gNeuronInput_I, buffers[d]->c_gFiredTable, buffers[d]->c_gFiredTableSizes, c_pNetGPU[d]->pNeuronNums[i+1]-c_pNetGPU[d]->pNeuronNums[i], c_pNetGPU[d]->pNeuronNums[i], time, &updateSize[d][c_pNetGPU[d]->pNTypes[i]]);
			}

			hipMemset(c_g_fired_n_num[d], 0, sizeof(int)*node_nets[d]._nodeNum);
		}
		hipDeviceSynchronize();
		//cudaDeliverNeurons(c_g_idx2index, c_g_cross_index2idx, c_g_global_cross_data, c_g_fired_n_num, node_nets[d]._nodeNum, allNeuronNum);
		//for (int i=0; i<node_nets[d]._nodeNum; i++) {
		//	int offset = i * node_nets[d]._nodeNum + node_nets[d]._nodeIdx; 
		//	copyFromGPU<int>(&(global_cross_data[offset]._fired_n_num), c_g_fired_n_num + i, 1);
		//	if (global_cross_data[offset]._fired_n_num > 0) {
		//		copyFromGPU<int>(global_cross_data[offset]._fired_n_idxs, c_g_global_cross_data + allNeuronNum * i, global_cross_data[offset]._fired_n_num);
		//	}
		//}

		for (int d=0; d<device_count; d++) {
			cudaDeliverNeurons<<<(c_pNetGPU[d]->pConnection->nNum+MAX_BLOCK_SIZE-1)/MAX_BLOCK_SIZE, MAX_BLOCK_SIZE>>>(c_pNetGPU[d]->pConnection, buffers[d]->c_gFiredTable, buffers[d]->c_gFiredTableSizes, c_g_idx2index[d], c_g_cross_index2idx[d], c_g_global_cross_data[d], c_g_fired_n_num[d], node_nets[d]._nodeNum, time);

			checkCudaErrors(hipMemcpy(gCrossDataGPU->_firedNum + node_nets[d]._nodeIdx * node_nets[d]._nodeNum, c_g_fired_n_num[d], sizeof(int)*node_nets[d]._nodeNum, hipMemcpyDeviceToHost));
		}

		for (int d=0; d<device_count; d++) {
			for (int i=0; i< node_nets[d]._nodeNum; i++) {
				int idx2i = node_nets[d]._nodeIdx * node_nets[d]._nodeNum + i;
				assert(gCrossDataGPU->_firedNum[idx2i] <= gCrossDataGPU->_maxNum[idx2i]);
				if (gCrossDataGPU->_firedNum[idx2i] > 0) {
					checkCudaErrors(hipMemcpyPeer(gCrossDataGPU->_firedArrays[idx2i], i, c_g_global_cross_data[d] + c_pNetGPU[d]->pConnection->nNum * i, node_nets[d]._nodeIdx, gCrossDataGPU->_firedNum[idx2i] * sizeof(int)));
				}
			}
		}

#ifdef LOG_DATA
		int *copySize = (int *)malloc(sizeof(int) * device_count);
		for (int d=0; d<device_count; d++) {
			int currentIdx = time%(c_pNetGPU[d]->pConnection->maxDelay+1);

			copyFromGPU<int>(&copySize[d], buffers[d]->c_gFiredTableSizes + currentIdx, 1);
			if (copySize[d] > 0) {
				copyFromGPU<int>(buffers[d]->c_neuronsFired, buffers[d]->c_gFiredTable + (c_pNetGPU[d]->pConnection->nNum*currentIdx), copySize[d]);
			}

			if (copy_idx[d] >= 0 && (c_pNetGPU[d]->pNeuronNums[copy_idx[d]+1]-c_pNetGPU[d]->pNeuronNums[copy_idx[d]]) > 0) {
				copyFromGPU<real>(c_vm[d], c_g_vm[d], c_pNetGPU[d]->pNeuronNums[copy_idx[d]+1]-c_pNetGPU[d]->pNeuronNums[copy_idx[d]]);
			}
		}
#endif

		for (int d=0; d<device_count; d++) {
			for (int i=0; i<c_pNetGPU[d]->sTypeNum; i++) {
				assert(c_pNetGPU[d]->pSynapseNums[i+1]-c_pNetGPU[d]->pSynapseNums[i] > 0);
				cudaUpdateType[c_pNetGPU[d]->pSTypes[i]](c_pNetGPU[d]->pConnection, c_pNetGPU[d]->ppSynapses[i], buffers[d]->c_gNeuronInput, buffers[d]->c_gNeuronInput_I, buffers[d]->c_gFiredTable, buffers[d]->c_gFiredTableSizes, c_pNetGPU[d]->pSynapseNums[i+1]-c_pNetGPU[d]->pSynapseNums[i], c_pNetGPU[d]->pSynapseNums[i], time, &updateSize[d][c_pNetGPU[d]->pSTypes[i]]);
			}
		}		
		hipDeviceSynchronize();

		//collectNeurons();
		//if (global_cross_data[dataIdx]._fired_n_num > 0) {
		//	copyToGPU(c_g_cross_id, global_cross_data[dataIdx]._fired_n_idxs, global_cross_data[dataIdx]._fired_n_num);
		//	addCrossNeurons(c_g_cross_id, global_cross_data[dataIdx]._fired_n_num);
		//}
		for (int d=0; d<device_count; d++) {
			for (int i=0; i< node_nets[d]._nodeNum; i++) {
				int i2idx = node_nets[d]._nodeIdx + node_nets[d]._nodeNum * i;
				if (gCrossDataGPU->_firedNum[i2idx] > 0) {
					int num = gCrossDataGPU->_firedNum[i2idx];
					cudaAddCrossNeurons<<<(num+MAX_BLOCK_SIZE-1)/MAX_BLOCK_SIZE, MAX_BLOCK_SIZE>>>(c_pNetGPU[d]->pConnection, buffers[d]->c_gFiredTable, buffers[d]->c_gFiredTableSizes, gCrossDataGPU->_firedArrays[i2idx], gCrossDataGPU->_firedNum[i2idx], time);
				}
			}
		}
		
#ifdef LOG_DATA
		for (int d=0; d<device_count; d++) {
			for (int i=0; i<copySize[d]; i++) {
				fprintf(log_file[d], "%d ", buffers[d]->c_neuronsFired[i]);
			}
			fprintf(log_file[d], "\n");

			for (int i=0; i<c_pNetGPU[d]->pNeuronNums[copy_idx[d]+1] - c_pNetGPU[d]->pNeuronNums[copy_idx[d]]; i++) {
				fprintf(v_file[d], "%.10lf \t", c_vm[d][i]);
			}
			fprintf(v_file[d], "\n");
		}
#endif

	}
	gettimeofday(&te, NULL);
	long seconds = te.tv_sec - ts.tv_sec;
	long hours = seconds/3600;
	seconds = seconds%3600;
	long minutes = seconds/60;
	seconds = seconds%60;
	long uSeconds = te.tv_usec - ts.tv_usec;
	if (uSeconds < 0) {
		uSeconds += 1000000;
		seconds = seconds - 1;
	}

	printf("Simulation finesed in %ld:%ld:%ld.%06lds\n", hours, minutes, seconds, uSeconds);

	for (int d=0; d<device_count; d++) {
		int nodeNeuronNum = c_pNetGPU[d]->pNeuronNums[c_pNetGPU[d]->nTypeNum];
		int *rate = (int*)malloc(sizeof(int)*nodeNeuronNum);
		copyFromGPU<int>(rate, buffers[d]->c_gFireCount, nodeNeuronNum);

		char fire_filename[512];
		sprintf(fire_filename, "fire_%d.count", node_nets[d]._nodeIdx); 
		FILE *rate_file = fopen(fire_filename, "w+");
		if (rate_file == NULL) {
			printf("Open file Sim.log failed\n");
			return -1;
		}

		for (int i=0; i<nodeNeuronNum; i++) {
			fprintf(rate_file, "%d \t", rate[i]);
		}
		free(rate);
		fclose(rate_file);
	}

	for (int d=0; d<device_count; d++) {
		fclose(log_file[d]);
		fclose(v_file[d]);

		free_buffers(buffers[d]);
		freeNetworkGPU(c_pNetGPU[d]);
	}

	return 0;
}
