#include "hip/hip_runtime.h"
/* This program is writen by qp09.
 * Sat October 24 2015
 */

#include <sys/time.h>
#include <stdio.h>
#include <assert.h>
#include <pthread.h>
#include <iostream>

#include "../utils/utils.h"
#include "../base/TypeFunc.h"
#include "../../msg_utils/helper/helper_gpu.h"
#include "../gpu_utils/runtime.h"
//#include "../gpu_utils/GBuffers.h"
#include "../net/Network.h"
#include "../neuron/lif/LIFData.h"

#include "MultiGPUSimulator.h"

using std::cout;
using std::endl;

pthread_barrier_t gpuCycleBarrier;

CrossThreadDataGPU * gCrossDataGPU;

MultiGPUSimulator::MultiGPUSimulator(Network *network, real dt) : Simulator(network, dt)
{
}

MultiGPUSimulator::~MultiGPUSimulator()
{
}

void *run_thread_gpu(void *para);

int MultiGPUSimulator::run(real time, FireInfo &log)
{
	int sim_cycle = round(time/_dt);
	reset();

	int device_count = 4;
	checkCudaErrors(hipGetDeviceCount(&device_count));
	assert(device_count != 0);
	for (int i=0; i<device_count; i++) {
		for (int j=0; j<device_count; j++) {
			if (i!=j) {
				int access = 0;
				checkCudaErrors(hipDeviceCanAccessPeer(&access, i, j));
				if (access == 1) {
					checkCudaErrors(hipSetDevice(i));
					checkCudaErrors(hipDeviceEnablePeerAccess(j, 0));
				}
			}
		}
	}
	checkCudaErrors(hipSetDevice(0));

	pthread_barrier_init(&gpuCycleBarrier, NULL, device_count);

	// MultiNetwork multiNet(_network, device_count);
	_network->set_node_num(device_count);

	SimInfo info(_dt);
	DistriNetwork *node_nets = _network->buildNetworks(info);
	assert(node_nets != NULL);
	gCrossDataGPU = _network->arrangeCrossGPUData();
	assert(gCrossDataGPU != NULL);

	pthread_t *thread_ids = (pthread_t *)malloc(sizeof(pthread_t) * device_count);
	assert(thread_ids != NULL);


	for (int i=0; i<device_count; i++) {
		node_nets[i]._simCycle = sim_cycle;
		node_nets[i]._nodeIdx = i;
		node_nets[i]._nodeNum = device_count;
		node_nets[i]._dt = _dt;


		int ret = pthread_create(&(thread_ids[i]), NULL, &run_thread_gpu, (void*)&(node_nets[i]));
		assert(ret == 0);
	}

	for (int i=0; i<device_count; i++) {
		pthread_join(thread_ids[i], NULL);
	}

	pthread_barrier_destroy(&gpuCycleBarrier);

	return 0;
}

void * run_thread_gpu(void *para) {
	DistriNetwork *network = (DistriNetwork*)para;

	char log_filename[512];
	sprintf(log_filename, "sim_%d.gpu.log", network->_nodeIdx); 
	FILE *log_file = fopen(log_filename, "w+");
	assert(log_file != NULL);

	char v_filename[512];
	sprintf(v_filename, "v_%d.gpu.log", network->_nodeIdx); 
	FILE *v_file = fopen(v_filename, "w+");
	assert(v_file != NULL);

	checkCudaErrors(hipSetDevice(network->_nodeIdx));

	GNetwork *pNetCPU = network->_network;
	GNetwork *c_pNetGPU = copyGNetworkToGPU(pNetCPU);

	int nTypeNum = c_pNetGPU->nTypeNum;
	int sTypeNum = c_pNetGPU->sTypeNum;
	int nodeNeuronNum = c_pNetGPU->pNeuronNums[nTypeNum];
	int allNeuronNum = pNetCPU->ppConnections[0]->nNum;
	int nodeSynapseNum = c_pNetGPU->pSynapseNums[sTypeNum];
	printf("Thread %d NeuronTypeNum: %d, SynapseTypeNum: %d\n", network->_nodeIdx, nTypeNum, sTypeNum);
	printf("Thread %d NeuronNum: %d, SynapseNum: %d\n", network->_nodeIdx, nodeNeuronNum, nodeSynapseNum);

	//int dataOffset = network->_nodeIdx * network->_nodeNum;
	//int dataIdx = network->_nodeIdx * network->_nodeNum + network->_nodeIdx;

	int maxDelay = pNetCPU->ppConnections[0]->maxDelay;
	int minDelay = pNetCPU->ppConnections[0]->minDelay;
	// int deltaDelay = maxDelay - minDelay;
	// int deltaDelay = pNetCPU->pConnection->maxDelay - pNetCPU->pConnection->minDelay;
	printf("Thread %d MaxDelay: %d MinDelay: %d\n", network->_nodeIdx, maxDelay,  minDelay);

	// init_connection<<<1, 1>>>(c_pNetGPU->pConnection);

	// GBuffers *buffers = alloc_buffers(allNeuronNum, nodeSynapseNum, pNetCPU->ppConnections[0]->maxDelay, network->_dt);
	Buffer buffer(pNetCPU->bufferOffsets[nTypeNum], allNeuronNum, maxDelay, network->_nodeIdx);
	Buffer *g_buffer = buffer._gpu_array;

	BlockSize *updateSize = getBlockSize(allNeuronNum, nodeSynapseNum);

#ifdef LOG_DATA
	real *c_vm = hostMalloc<real>(nodeNeuronNum);
	int life_idx = getIndex(c_pNetGPU->pNTypes, nTypeNum, LIF);
	int copy_idx = -1;
	real *c_g_vm = NULL;

	if (life_idx >= 0) {
		LIFData *c_g_lif = copyFromGPU<LIFData>(static_cast<LIFData *>(c_pNetGPU->ppNeurons[life_idx]), 1);
		c_g_vm = c_g_lif->pV_m;
		copy_idx = life_idx;
	} else {
	}
#endif

	for (int i=0; i<nTypeNum; i++) {
		cout << "Thread " << network->_nodeIdx << " " << c_pNetGPU->pNTypes[i] << ": <<<" << updateSize[c_pNetGPU->pNTypes[i]].gridSize << ", " << updateSize[c_pNetGPU->pNTypes[i]].blockSize << ">>>" << endl;
	}
	for (int i=0; i<sTypeNum; i++) {
		cout << "Thread " << network->_nodeIdx << " " << c_pNetGPU->pSTypes[i] << ": <<<" << updateSize[c_pNetGPU->pSTypes[i]].gridSize << ", " << updateSize[c_pNetGPU->pSTypes[i]].blockSize << ">>>" << endl;
	}

	//int * c_g_cross_id = gpuMalloc<int>(global_cross_data[dataIdx]._max_n_num); 
	CrossNodeMap *cnm_gpu = to_gpu(network->_crossnodeMap);

	// integer_t * c_g_idx2index = copyToGPU(network->_crossnodeMap->_idx2index, allNeuronNum);
	// integer_t * c_g_cross_index2idx = copyToGPU(network->_crossnodeMap->_crossnodeIndex2idx, network->_crossnodeMap->_crossSize);
	uinteger_t * c_g_global_cross_data = gpuMalloc<uinteger_t>(allNeuronNum * network->_nodeNum);
	uinteger_t * c_g_fired_n_num = gpuMalloc<uinteger_t>(network->_nodeNum);

	vector<int> firedInfo;
	struct timeval ts, te;
	//struct timeval t0, t1, t2, t3, t4, t5,/* t6,*/ t7, t8, t9;
	//double barrier1_time = 0, gpu_cpy_time = 0, peer_cpy_time = 0, barrier2_time=0, copy_time = 0;
	gettimeofday(&ts, NULL);
	for (int time=0; time<network->_simCycle; time++) {
		update_time<<<1, 1>>>(g_buffer->_fired_sizes, maxDelay, time);

		for (int i=0; i<nTypeNum; i++) {
			assert(c_pNetGPU->pNeuronNums[i+1]-c_pNetGPU->pNeuronNums[i] > 0);
			cudaUpdateType[c_pNetGPU->pNTypes[i]](c_pNetGPU->ppConnections[i], c_pNetGPU->ppNeurons[i], g_buffer->_data, g_buffer->_fire_table, g_buffer->_fired_sizes, allNeuronNum, c_pNetGPU->pNeuronNums[i+1]-c_pNetGPU->pNeuronNums[i], c_pNetGPU->pNeuronNums[i], time, &updateSize[c_pNetGPU->pNTypes[i]]);
		}

		//gettimeofday(&t0, NULL);
		pthread_barrier_wait(&gpuCycleBarrier);
		//gettimeofday(&t1, NULL);
		//barrier1_time += (t1.tv_sec - t0.tv_sec) + (t1.tv_usec - t0.tv_usec)/1000000.0;
		hipMemset(c_g_fired_n_num, 0, sizeof(int)*network->_nodeNum);
		//hipDeviceSynchronize();
		//gettimeofday(&t2, NULL);
		//cudaDeliverNeurons(c_g_idx2index, c_g_cross_index2idx, c_g_global_cross_data, c_g_fired_n_num, network->_nodeNum, allNeuronNum);
		//for (int i=0; i<network->_nodeNum; i++) {
		//	int offset = i * network->_nodeNum + network->_nodeIdx; 
		//	copyFromGPU<int>(&(global_cross_data[offset]._fired_n_num), c_g_fired_n_num + i, 1);
		//	if (global_cross_data[offset]._fired_n_num > 0) {
		//		copyFromGPU<int>(global_cross_data[offset]._fired_n_idxs, c_g_global_cross_data + allNeuronNum * i, global_cross_data[offset]._fired_n_num);
		//	}
		//}

		cudaDeliverNeurons<<<(allNeuronNum+MAX_BLOCK_SIZE-1)/MAX_BLOCK_SIZE, MAX_BLOCK_SIZE>>>(g_buffer->_fire_table, g_buffer->_fired_sizes, cnm_gpu->_idx2index, cnm_gpu->_crossnodeIndex2idx, c_g_global_cross_data, c_g_fired_n_num, maxDelay, network->_nodeNum, time);

		copyFromGPU(gCrossDataGPU->_firedNum + network->_nodeIdx * network->_nodeNum, c_g_fired_n_num, network->_nodeNum);
		// checkCudaErrors(hipMemcpy(gCrossDataGPU->_firedNum + network->_nodeIdx * network->_nodeNum, c_g_fired_n_num, sizeof(int)*network->_nodeNum, hipMemcpyDeviceToHost));
		//gettimeofday(&t3, NULL);

		for (int i=0; i< network->_nodeNum; i++) {
			int idx2i = network->_nodeIdx * network->_nodeNum + i;
			assert(gCrossDataGPU->_firedNum[idx2i] <= gCrossDataGPU->_maxNum[idx2i]);
			if (gCrossDataGPU->_firedNum[idx2i] > 0) {
				gpuMemcpyPeer(gCrossDataGPU->_firedArrays[idx2i], i, c_g_global_cross_data + allNeuronNum * i, network->_nodeIdx, gCrossDataGPU->_firedNum[idx2i]);
			}
		}
		//gettimeofday(&t7, NULL);

		//gpu_cpy_time += (t3.tv_sec - t2.tv_sec) + (t3.tv_usec - t2.tv_usec)/1000000.0;
		//peer_cpy_time += (t7.tv_sec - t3.tv_sec) + (t7.tv_usec - t3.tv_usec)/1000000.0;

#ifdef LOG_DATA
		int currentIdx = time%(maxDelay+1);

		uinteger_t copySize = 0;
		copyFromGPU(&copySize, g_buffer->_fired_sizes + currentIdx, 1);
		assert(copySize <= allNeuronNum);
		if (copySize > 0) {
			copyFromGPU(buffer._fire_table, g_buffer->_fire_table + (allNeuronNum*currentIdx), copySize);
		}

		if (copy_idx >= 0 && (c_pNetGPU->pNeuronNums[copy_idx+1]-c_pNetGPU->pNeuronNums[copy_idx]) > 0) {
			copyFromGPU(c_vm, c_g_vm, c_pNetGPU->pNeuronNums[copy_idx+1]-c_pNetGPU->pNeuronNums[copy_idx]);
		}
#endif

		for (int i=0; i<sTypeNum; i++) {
			assert(c_pNetGPU->pSynapseNums[i+1]-c_pNetGPU->pSynapseNums[i] > 0);
			cudaUpdateType[c_pNetGPU->pSTypes[i]](c_pNetGPU->ppConnections[i], c_pNetGPU->ppSynapses[i], g_buffer->_data, g_buffer->_fire_table, g_buffer->_fired_sizes, allNeuronNum, c_pNetGPU->pSynapseNums[i+1]-c_pNetGPU->pSynapseNums[i], c_pNetGPU->pSynapseNums[i], time, &updateSize[c_pNetGPU->pSTypes[i]]);
		}
		//hipDeviceSynchronize();

		//gettimeofday(&t4, NULL);
		pthread_barrier_wait(&gpuCycleBarrier);
		//gettimeofday(&t5, NULL);
		//barrier2_time += (t5.tv_sec - t4.tv_sec) + (t5.tv_usec - t4.tv_usec)/1000000.0;

		//gettimeofday(&t6, NULL);
		//collectNeurons();
		//gettimeofday(&t7, NULL);
		//cpu_cpy_time += (t7.tv_sec - t6.tv_sec) + (t7.tv_usec - t6.tv_usec)/1000000.0;
		
		//gettimeofday(&t8, NULL);
		//if (global_cross_data[dataIdx]._fired_n_num > 0) {
		//	copyToGPU(c_g_cross_id, global_cross_data[dataIdx]._fired_n_idxs, global_cross_data[dataIdx]._fired_n_num);
		//	addCrossNeurons(c_g_cross_id, global_cross_data[dataIdx]._fired_n_num);
		//}
		for (int i=0; i< network->_nodeNum; i++) {
			int i2idx = network->_nodeIdx + network->_nodeNum * i;
			if (gCrossDataGPU->_firedNum[i2idx] > 0) {
				size_t num = gCrossDataGPU->_firedNum[i2idx];
				cudaAddCrossNeurons<<<(num+MAX_BLOCK_SIZE-1)/MAX_BLOCK_SIZE, MAX_BLOCK_SIZE>>>(g_buffer->_fire_table, g_buffer->_fired_sizes, gCrossDataGPU->_firedArrays[i2idx], gCrossDataGPU->_firedNum[i2idx], maxDelay, time);
			}
		}
		
		//gettimeofday(&t9, NULL);
		//copy_time += (t9.tv_sec - t8.tv_sec) + (t9.tv_usec - t8.tv_usec)/1000000.0;

#ifdef LOG_DATA
		for (int i=0; i<copySize; i++) {
			fprintf(log_file, "%d ", buffer._fire_table[i]);
		}
		fprintf(log_file, "\n");

		for (int i=0; i<c_pNetGPU->pNeuronNums[copy_idx+1] - c_pNetGPU->pNeuronNums[copy_idx]; i++) {
			fprintf(v_file, "%.10lf \t", c_vm[i]);
		}
		fprintf(v_file, "\n");
#endif

		//pthread_barrier_wait(&gpuCycleBarrier);
	}
	pthread_barrier_wait(&gpuCycleBarrier);
	gettimeofday(&te, NULL);
	long seconds = te.tv_sec - ts.tv_sec;
	long hours = seconds/3600;
	seconds = seconds%3600;
	long minutes = seconds/60;
	seconds = seconds%60;
	long uSeconds = te.tv_usec - ts.tv_usec;
	if (uSeconds < 0) {
		uSeconds += 1000000;
		seconds = seconds - 1;
	}

	printf("Thread %d Simulation finesed in %ld:%ld:%ld.%06lds\n", network->_nodeIdx, hours, minutes, seconds, uSeconds);
	//printf("Thread %d cost : barrier1 %lf, DtoH %lf, DtoD %lf, barrier2 %lf, HtoD %lf\n", network->_nodeIdx, barrier1_time, gpu_cpy_time, peer_cpy_time, barrier2_time, copy_time);

	char fire_filename[512];
	sprintf(fire_filename, "gpu_%d", network->_nodeIdx); 

	for (int i=0; i<nTypeNum; i++) {
		cudaLogRateNeuron[pNetCPU->pNTypes[i]](pNetCPU->ppNeurons[i], c_pNetGPU->ppNeurons[i],  fire_filename);
	}

	fclose(log_file);
	fclose(v_file);

	// free_buffers(buffers);
	freeGNetworkGPU(c_pNetGPU);

	return NULL;
}

