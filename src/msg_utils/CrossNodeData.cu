
#include <assert.h>

#include "../third_party/cuda/hip/hip_runtime_api.h"

#include "CrossNodeData.h"

CrossNodeData * copyCNDtoGPU(CrossNodeData *data)
{
	// TODO correct size;
	CrossNodeData *gpu = (CrossNodeData *)malloc(sizeof(CrossNodeData));
	assert(gpu != NULL);

	gpu->_node_num = data->_node_num;
	gpu->_delay = data->_delay;

	int size = data->_node_num * data->_delay;
	int num_p_1 = data->_node_num + 1;

	checkCudaErrors(hipMalloc((void**)&(gpu->_recv_offset), sizeof(int)*num_p_1));
	checkCudaErrors(hipMemcpy(gpu->_recv_offset, data->_recv_offset, sizeof(int)*num_p_1, hipMemcpyHostToDevice));


	checkCudaErrors(hipMalloc((void**)&(gpu->_recv_num), sizeof(int)*(size)));
	checkCudaErrors(hipMemset(gpu->_recv_num, 0, sizeof(int)*(size)));

	checkCudaErrors(hipMalloc((void**)&(gpu->_recv_data), sizeof(int)*(data->_recv_offset[size])));
	checkCudaErrors(hipMemset(gpu->_recv_data, 0, sizeof(int)*(data->_recv_offset[size])));

	checkCudaErrors(hipMalloc((void**)&(gpu->_send_offset), sizeof(int)*(size + 1)));
	checkCudaErrors(hipMemcpy(gpu->_send_offset, data->_send_offset, sizeof(int)*(size + 1), hipMemcpyHostToDevice));


	checkCudaErrors(hipMalloc((void**)&(gpu->_send_num), sizeof(int)*(size)));
	checkCudaErrors(hipMemset(gpu->_send_num, 0, sizeof(int)*(size)));

	checkCudaErrors(hipMalloc((void**)&(gpu->_send_data), sizeof(int)*(data->_send_offset[size])));
	checkCudaErrors(hipMemset(gpu->_send_data, 0, sizeof(int)*(data->_send_offset[size])));

	return gpu;
}

int freeCNDGPU(CrossNodeData *data) 
{
	hipFree(data->_recv_offset);
	hipFree(data->_recv_num);
	hipFree(data->_recv_data);

	hipFree(data->_send_offset);
	hipFree(data->_send_num);
	hipFree(data->_send_data);

	data->_node_num = 0;
	data->_delay = 0;
	free(data);
	data = NULL;
	return 0;
}
