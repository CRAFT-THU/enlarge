
#include <assert.h>

#include "../third_party/cuda/hip/hip_runtime_api.h"

#include "CrossNodeData.h"

CrossNodeData * copyCNDtoGPU(CrossNodeData *data)
{
	// TODO correct size;
	CrossNodeData *gpu = (CrossNodeData *)malloc(sizeof(CrossNodeData));
	assert(gpu != NULL);

	gpu->_node_num = data->_node_num;
	gpu->_min_delay = data->_min_delay;

	int num = data->_node_num;
	int size = data->_node_num * data->_min_delay;
	int num_p_1 = data->_node_num + 1;

	checkCudaErrors(hipMalloc((void**)&(gpu->_recv_offset), sizeof(int)*num_p_1));
	checkCudaErrors(hipMemcpy(gpu->_recv_offset, data->_recv_offset, sizeof(int)*num_p_1, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(gpu->_recv_start), sizeof(int)*(size+num)));
	checkCudaErrors(hipMemcpy(gpu->_recv_start, data->_recv_start, sizeof(int)*(size+num), hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(gpu->_recv_num), sizeof(int)*num));
	checkCudaErrors(hipMemset(gpu->_recv_num, 0, sizeof(int)*num));

	checkCudaErrors(hipMalloc((void**)&(gpu->_recv_data), sizeof(int)*(data->_recv_offset[num])));
	checkCudaErrors(hipMemset(gpu->_recv_data, 0, sizeof(int)*(data->_recv_offset[num])));

	checkCudaErrors(hipMalloc((void**)&(gpu->_send_offset), sizeof(int)*num_p_1));
	checkCudaErrors(hipMemcpy(gpu->_send_offset, data->_send_offset, sizeof(int)*num_p_1, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(gpu->_send_start), sizeof(int)*(size+num)));
	checkCudaErrors(hipMemcpy(gpu->_send_start, data->_send_start, sizeof(int)*(size+num), hipMemcpyHostToDevice));


	checkCudaErrors(hipMalloc((void**)&(gpu->_send_num), sizeof(int)*num));
	checkCudaErrors(hipMemset(gpu->_send_num, 0, sizeof(int)*num));

	checkCudaErrors(hipMalloc((void**)&(gpu->_send_data), sizeof(int)*(data->_send_offset[num])));
	checkCudaErrors(hipMemset(gpu->_send_data, 0, sizeof(int)*(data->_send_offset[num])));

	return gpu;
}

int freeCNDGPU(CrossNodeData *data) 
{
	hipFree(data->_recv_offset);
	hipFree(data->_recv_start);
	hipFree(data->_recv_num);
	hipFree(data->_recv_data);

	hipFree(data->_send_offset);
	hipFree(data->_send_start);
	hipFree(data->_send_num);
	hipFree(data->_send_data);

	data->_node_num = 0;
	data->_min_delay = 0;
	free(data);
	data = NULL;
	return 0;
}
