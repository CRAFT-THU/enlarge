#include "hip/hip_runtime.h"

#include "LIFNmdaData.h"

#include "../../../msg_utils/helper/helper_gpu.h"
#include "../../gpu_utils/runtime.h"
#include "../../net/Connection.h"

// __global__ void find_lif_neuron(LIFData *data, real * currentE, real * currentI, int num, int offset)
// {
// 	__shared__ uinteger_t tActiveTable[MAX_BLOCK_SIZE];
// 	__shared__ volatile uinteger_t activeCnt;
// 
// 	if (threadIdx.x == 0) {
// 		activeCnt = 0;
// 	}
// 	__syncthreads();
// 
// 	uinteger_t tid = blockIdx.x * blockDim.x + threadIdx.x;
// 	for (uinteger_t idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
// 		//bool actived = false;
// 		int testLoc = 0;
// 		bool actived = data->pRefracStep[idx] <= 0;
// 		if (actived) {
// 			testLoc = atomicAdd((int*)&activeCnt, 1);
// 			if (testLoc < MAX_BLOCK_SIZE) {
// 				tActiveTable[testLoc] = idx;
// 				actived = false;
// 			}
// 		} else {
// 			currentE[offset + idx] = 0;
// 			currentI[offset + idx] = 0;
// 			data->pRefracStep[idx] = data->pRefracStep[idx] - 1;
// 		}
// 		__syncthreads();
// 
// 		if (activeCnt >= MAX_BLOCK_SIZE) {
// 			commit2globalTable(tActiveTable, MAX_BLOCK_SIZE, gActiveTable, &gActiveTableSize, 0);
// 			if (threadIdx.x == 0) {
// 				activeCnt = 0;
// 			}
// 		}
// 		__syncthreads();
// 
// 		if (actived) {
// 			testLoc = atomicAdd((int*)&activeCnt, 1);
// 			if (testLoc < MAX_BLOCK_SIZE) {
// 				tActiveTable[testLoc] = idx;
// 				actived = false;
// 			}
// 		}
// 		__syncthreads();
// 
// 		if (activeCnt >= MAX_BLOCK_SIZE) {
// 			commit2globalTable(tActiveTable, MAX_BLOCK_SIZE, gActiveTable, &gActiveTableSize, 0);
// 			if (threadIdx.x == 0) {
// 				activeCnt = 0;
// 			}
// 		}
// 		__syncthreads();
// 
// 		if (activeCnt > 0) {
// 			commit2globalTable(tActiveTable, activeCnt, gActiveTable, &gActiveTableSize, 0);
// 			if (threadIdx.x == 0) {
// 				activeCnt = 0;
// 			}
// 		}
// 		__syncthreads();
// 	}
// }

// __global__ void update_lif_neuron(Connection *connection, LIFData *data, real *currentE, real *currentI, int *firedTable, int *firedTableSizes, int num, int offset, int time)
// {
// 	int currentIdx = time % (connection->maxDelay+1);
// 	__shared__ int fire_table_t[MAX_BLOCK_SIZE];
// 	__shared__ volatile int fire_cnt;
// 	if (threadIdx.x == 0) {
// 		fire_cnt = 0;
// 	}
// 	__syncthreads();
// 
// 	int tid = blockIdx.x * blockDim.x + threadIdx.x;
// 	for (int idx = tid; idx < gActiveTableSize; idx +=blockDim.x*gridDim.x) {
// 		bool fired = false;
// 		int testLoc = 0;
// 
// 		int nid = gActiveTable[idx];
// 		int gnid = offset + nid; 
// 
// 		//real I = sqrtf(data->pCe[nid]) * data->pI_e[nid] + sqrtf(data->pCi[nid]) * data->pI_i[nid] + data->p_i_tmp[nid];
// 
// 		//real I = currentE[gnid] + data->p_i_tmp[nid];
// 		//data->pV_m[nid] = data->pV_m[nid] * data->p_C1[nid] + data->p_C2[nid] * I;
// 
// 		data->pV_m[nid] = data->pC_m[nid] * data->pV_m[nid] + data->pV_tmp[nid] + data->pI_e[nid] * data->pC_e[nid] + data->pI_i[nid] * data->pC_i[nid];
// 
// 		//data->p_i_syn[nid] = 0;
// 
// 		data->pI_e[nid] *= data->pCe[nid];
// 		data->pI_i[nid] *= data->pCi[nid];
// 
// 		fired = data->pV_m[nid] >= data->pV_thresh[nid];
// 
// 		gFireCount[gnid] += fired;
// 
// 		if (fired) {
// 			testLoc = atomicAdd((int*)&fire_cnt, 1);
// 			if (testLoc < MAX_BLOCK_SIZE) {
// 				fire_table_t[testLoc] = gnid;
// 				fired = false;
// 			}
// 
// 			data->pRefracStep[nid] = data->pRefracTime[nid] - 1;
// 			data->pV_m[nid] = data->pV_reset[nid];
// 		} else {
// 			gXInput[gnid] += currentE[gnid] + currentI[gnid];
// 			data->pI_e[nid] += currentE[gnid];
// 			data->pI_i[nid] += currentI[gnid];
// 		}
// 
// 		currentE[gnid] = 0;
// 		currentI[gnid] = 0;
// 
// 		__syncthreads();
// 		if (fire_cnt >= MAX_BLOCK_SIZE) {
// 			commit2globalTable(fire_table_t, MAX_BLOCK_SIZE, firedTable, &firedTableSizes[currentIdx], gFiredTableCap*currentIdx);
// 			if (threadIdx.x == 0) {
// 				fire_cnt = 0;
// 			}
// 		}
// 
// 		__syncthreads();
// 
// 		if (fired) {
// 			testLoc = atomicAdd((int*)&fire_cnt, 1);
// 			if (testLoc < MAX_BLOCK_SIZE) {
// 				fire_table_t[testLoc] = gnid;
// 				fired = false;
// 			}
// 		}
// 		__syncthreads();
// 		if (fire_cnt >= MAX_BLOCK_SIZE) {
// 			commit2globalTable(fire_table_t, MAX_BLOCK_SIZE, firedTable, &firedTableSizes[currentIdx], gFiredTableCap*currentIdx);
// 			if (threadIdx.x == 0) {
// 				fire_cnt = 0;
// 			}
// 		}
// 		__syncthreads();
// 
// 		if (fire_cnt > 0) {
// 			commit2globalTable(fire_table_t, fire_cnt, firedTable, &firedTableSizes[currentIdx], gFiredTableCap*currentIdx);
// 			if (threadIdx.x == 0) {
// 				fire_cnt = 0;
// 			}
// 		}
// 
// 	}
// 	//__syncthreads();
// 	//if (threadIdx.x == 0 && blockIdx.x == 0) {
// 	//	gActiveTableSize = 0;
// 	//}
// }

__global__ void update_all_lif_nmda_neuron(Connection *connection, LIFNmdaData *data, real *buffer, uinteger_t *firedTable, uinteger_t *firedTableSizes, size_t firedTableCap, size_t num, size_t offset, int time)
// __global__ void update_all_lif_neuron(LIFData *data, int num, int offset, int time)
{
	int currentIdx = time % (connection->maxDelay + 1);
	__shared__ uinteger_t fire_table_t[MAX_BLOCK_SIZE];
	__shared__ volatile uinteger_t fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}

	__syncthreads();

	size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (size_t idx = 0; idx < num; idx +=blockDim.x*gridDim.x) {
		bool fired = false;
		uinteger_t testLoc = 0;

		size_t nid = idx + tid;
		size_t gnid = offset + nid; 
		if (nid < num) {
			bool actived = data->pRefracStep[nid] <= 0;

			if (actived) {
                // ! 先更新电流、电压再判断是否发放
                real V = data->pV[nid];
                data->pI[nid] += buffer[gnid] * (data->pE[nid] - V) / (1 + data->pM_c[nid] * exp(-0.062 * V));
				data->pV[nid] = (1 - data->pC_m[nid]) * data->pV[nid] + data->pC_m[nid] * (data->pV_tmp[nid] + data->pR[nid] * data->pI[nid]);

				fired = data->pV[nid] >= data->pV_thresh[nid];

				data->_fire_count[gnid] += fired;

				if (fired) {
					testLoc = atomicAdd((uinteger_t *)&fire_cnt, 1);
					if (testLoc < MAX_BLOCK_SIZE) {
						fire_table_t[testLoc] = gnid;
						fired = false;
					}

					// data->pRefracStep[nid] = data->pRefracTime[nid] - 1; // ! 按需求严格打 time次拍 后再恢复
                    data->pRefracStep[nid] = data->pRefracTime[nid];
					data->pV[nid] = data->pV_reset[nid];
				}
			} else {
				data->pRefracStep[nid] -= 1;
			}

			buffer[gnid] = 0;
			buffer[gnid+num] = 0;
		}

		__syncthreads();
		if (fire_cnt >= MAX_BLOCK_SIZE) {
			commit2globalTable(fire_table_t, static_cast<uinteger_t>(MAX_BLOCK_SIZE), firedTable, &firedTableSizes[currentIdx], static_cast<uinteger_t>(firedTableCap*currentIdx));
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}

		__syncthreads();

		if (fired) {
			testLoc = atomicAdd((uinteger_t*)&fire_cnt, 1);
			if (testLoc < MAX_BLOCK_SIZE) {
				fire_table_t[testLoc] = gnid;
				fired = false;
			}
		}
		__syncthreads();
		if (fire_cnt >= MAX_BLOCK_SIZE) {
			commit2globalTable(fire_table_t, static_cast<uinteger_t>(MAX_BLOCK_SIZE), firedTable, &firedTableSizes[currentIdx], static_cast<uinteger_t>(firedTableCap*currentIdx));
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
		__syncthreads();

	}
	if (fire_cnt > 0) {
		commit2globalTable(fire_table_t, fire_cnt, firedTable, &firedTableSizes[currentIdx], static_cast<uinteger_t>(firedTableCap*currentIdx));
		if (threadIdx.x == 0) {
			fire_cnt = 0;
		}
	}
	__syncthreads();
}

// __global__ void update_dense_lif_neuron(Connection *connection, LIFData *data, real *buffer, int *firedTable, int *firedTableSizes, int firedTableCap, int num, int offset, int time)
// {
// 	//__shared__ int fire_table_t[MAX_BLOCK_SIZE];
// 	//__shared__ volatile int fire_cnt;
// 
// 	//if (threadIdx.x == 0) {
// 	//	fire_cnt = 0;
// 	//}
// 	//__syncthreads();
// 
// 	int tid = blockIdx.x * blockDim.x + threadIdx.x;
// 	int currentIdx = time % (connection->maxDelay+1);
// 	for (int idx = tid; idx < num; idx +=blockDim.x*gridDim.x) {
// 		//bool fired = false;
// 		//int testLoc = 0;
// 
// 		int nid = idx;
// 		int gnid = offset + idx; 
// 		bool actived = data->pRefracStep[idx] <= 0;
// 
// 		if (actived) {
// 			data->pV_m[nid] = data->pC_m[nid] * data->pV_m[nid] + data->pV_tmp[nid] + data->pI_e[nid] * data->pC_e[nid] + data->pI_i[nid] * data->pC_i[nid];
// 
// 			data->pI_e[nid] *= data->pCe[nid];
// 			data->pI_i[nid] *= data->pCi[nid];
// 
// 			bool fired = data->pV_m[nid] >= data->pV_thresh[nid];
// 
// 			firedTable[firedTableCap*currentIdx + gnid] = fired;
// 
// 			data->_fire_count[gnid] += fired;
// 
// 			if (fired) {
// 				data->pRefracStep[nid] = data->pRefracTime[nid] - 1;
// 				data->pV_m[nid] = data->pV_reset[nid];
// 
// 			} else {
// 				// gXInput[gnid] += currentE[gnid] + currentI[gnid];
// 				data->pI_e[nid] += buffer[gnid];
// 				data->pI_i[nid] += buffer[gnid+num];
// 				//real input = 0, input_I = 0;
// 				//for (int i=data->p_start_E[nid]; i<data->p_start_I[nid]; i++) {
// 				//	input += currentE[i];
// 				//}
// 				//for (int i=data->p_start_I[nid]; i<data->p_end[nid]; i++) {
// 				//	input_I += currentE[i];
// 				//}
// 				//data->pI_e[nid] += input;
// 				//data->pI_i[nid] += input_I;
// 				//gXInput[gnid] += input + input_I;
// 			}
// 
// 		} else {
// 			data->pRefracStep[idx] = data->pRefracStep[idx] - 1;
// 			firedTable[firedTableCap*currentIdx + gnid] = 0;
// 		}
// 		buffer[gnid] = 0;
// 		buffer[gnid+num] = 0;
// 	}
// 	__syncthreads();
// }

void cudaUpdateLIFNmda(Connection *conn, void *data, real *buffer, uinteger_t *firedTable, uinteger_t *firedTableSizes, size_t firedTableCap, size_t num, size_t offset, int time, BlockSize *pSize)
{
	// find_lif_neuron<<<pSize->gridSize, pSize->blockSize>>>((LIFData*)data, currentE, currentI, num, offset);
	// update_lif_neuron<<<pSize->gridSize, pSize->blockSize>>>(conn, (LIFData*)data, currentE, currentI, firedTable, firedTableSizes, num, offset, time);
	update_all_lif_nmda_neuron<<<pSize->gridSize, pSize->blockSize>>>(conn, (LIFNmdaData*)data, buffer, firedTable, firedTableSizes, firedTableCap, num, offset, time);

}
