#include "hip/hip_runtime.h"

#include "NMDAData.h"

#include "../../../msg_utils/helper/helper_gpu.h"
#include "../../gpu_utils/runtime.h"
#include "../../net/Connection.h"

// __global__ void find_nmda_neuron(NMDAData *data, real * currentE, real * currentI, int num, int offset)
// {
// 	__shared__ uinteger_t tActiveTable[MAX_BLOCK_SIZE];
// 	__shared__ volatile uinteger_t activeCnt;
// 
// 	if (threadIdx.x == 0) {
// 		activeCnt = 0;
// 	}
// 	__syncthreads();
// 
// 	uinteger_t tid = blockIdx.x * blockDim.x + threadIdx.x;
// 	for (uinteger_t idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
// 		//bool actived = false;
// 		int testLoc = 0;
// 		bool actived = data->pRefracStep[idx] <= 0;
// 		if (actived) {
// 			testLoc = atomicAdd((int*)&activeCnt, 1);
// 			if (testLoc < MAX_BLOCK_SIZE) {
// 				tActiveTable[testLoc] = idx;
// 				actived = false;
// 			}
// 		} else {
// 			currentE[offset + idx] = 0;
// 			currentI[offset + idx] = 0;
// 			data->pRefracStep[idx] = data->pRefracStep[idx] - 1;
// 		}
// 		__syncthreads();
// 
// 		if (activeCnt >= MAX_BLOCK_SIZE) {
// 			commit2globalTable(tActiveTable, MAX_BLOCK_SIZE, gActiveTable, &gActiveTableSize, 0);
// 			if (threadIdx.x == 0) {
// 				activeCnt = 0;
// 			}
// 		}
// 		__syncthreads();
// 
// 		if (actived) {
// 			testLoc = atomicAdd((int*)&activeCnt, 1);
// 			if (testLoc < MAX_BLOCK_SIZE) {
// 				tActiveTable[testLoc] = idx;
// 				actived = false;
// 			}
// 		}
// 		__syncthreads();
// 
// 		if (activeCnt >= MAX_BLOCK_SIZE) {
// 			commit2globalTable(tActiveTable, MAX_BLOCK_SIZE, gActiveTable, &gActiveTableSize, 0);
// 			if (threadIdx.x == 0) {
// 				activeCnt = 0;
// 			}
// 		}
// 		__syncthreads();
// 
// 		if (activeCnt > 0) {
// 			commit2globalTable(tActiveTable, activeCnt, gActiveTable, &gActiveTableSize, 0);
// 			if (threadIdx.x == 0) {
// 				activeCnt = 0;
// 			}
// 		}
// 		__syncthreads();
// 	}
// }

// __global__ void update_nmda_neuron(Connection *connection, NMDAData *data, real *currentE, real *currentI, int *firedTable, int *firedTableSizes, int num, int offset, int time)
// {
// 	int currentIdx = time % (connection->maxDelay+1);
// 	__shared__ int fire_table_t[MAX_BLOCK_SIZE];
// 	__shared__ volatile int fire_cnt;
// 	if (threadIdx.x == 0) {
// 		fire_cnt = 0;
// 	}
// 	__syncthreads();
// 
// 	int tid = blockIdx.x * blockDim.x + threadIdx.x;
// 	for (int idx = tid; idx < gActiveTableSize; idx +=blockDim.x*gridDim.x) {
// 		bool fired = false;
// 		int testLoc = 0;
// 
// 		int nid = gActiveTable[idx];
// 		int gnid = offset + nid; 
// 
// 		//real I = sqrtf(data->pCe[nid]) * data->pI_e[nid] + sqrtf(data->pCi[nid]) * data->pI_i[nid] + data->p_i_tmp[nid];
// 
// 		//real I = currentE[gnid] + data->p_i_tmp[nid];
// 		//data->pV_m[nid] = data->pV_m[nid] * data->p_C1[nid] + data->p_C2[nid] * I;
// 
// 		data->pV_m[nid] = data->pC_m[nid] * data->pV_m[nid] + data->pV_tmp[nid] + data->pI_e[nid] * data->pC_e[nid] + data->pI_i[nid] * data->pC_i[nid];
// 
// 		//data->p_i_syn[nid] = 0;
// 
// 		data->pI_e[nid] *= data->pCe[nid];
// 		data->pI_i[nid] *= data->pCi[nid];
// 
// 		fired = data->pV_m[nid] >= data->pV_thresh[nid];
// 
// 		gFireCount[gnid] += fired;
// 
// 		if (fired) {
// 			testLoc = atomicAdd((int*)&fire_cnt, 1);
// 			if (testLoc < MAX_BLOCK_SIZE) {
// 				fire_table_t[testLoc] = gnid;
// 				fired = false;
// 			}
// 
// 			data->pRefracStep[nid] = data->pRefracTime[nid] - 1;
// 			data->pV_m[nid] = data->pV_reset[nid];
// 		} else {
// 			gXInput[gnid] += currentE[gnid] + currentI[gnid];
// 			data->pI_e[nid] += currentE[gnid];
// 			data->pI_i[nid] += currentI[gnid];
// 		}
// 
// 		currentE[gnid] = 0;
// 		currentI[gnid] = 0;
// 
// 		__syncthreads();
// 		if (fire_cnt >= MAX_BLOCK_SIZE) {
// 			commit2globalTable(fire_table_t, MAX_BLOCK_SIZE, firedTable, &firedTableSizes[currentIdx], gFiredTableCap*currentIdx);
// 			if (threadIdx.x == 0) {
// 				fire_cnt = 0;
// 			}
// 		}
// 
// 		__syncthreads();
// 
// 		if (fired) {
// 			testLoc = atomicAdd((int*)&fire_cnt, 1);
// 			if (testLoc < MAX_BLOCK_SIZE) {
// 				fire_table_t[testLoc] = gnid;
// 				fired = false;
// 			}
// 		}
// 		__syncthreads();
// 		if (fire_cnt >= MAX_BLOCK_SIZE) {
// 			commit2globalTable(fire_table_t, MAX_BLOCK_SIZE, firedTable, &firedTableSizes[currentIdx], gFiredTableCap*currentIdx);
// 			if (threadIdx.x == 0) {
// 				fire_cnt = 0;
// 			}
// 		}
// 		__syncthreads();
// 
// 		if (fire_cnt > 0) {
// 			commit2globalTable(fire_table_t, fire_cnt, firedTable, &firedTableSizes[currentIdx], gFiredTableCap*currentIdx);
// 			if (threadIdx.x == 0) {
// 				fire_cnt = 0;
// 			}
// 		}
// 
// 	}
// 	//__syncthreads();
// 	//if (threadIdx.x == 0 && blockIdx.x == 0) {
// 	//	gActiveTableSize = 0;
// 	//}
// }

__global__ void update_all_nmda_neuron(Connection *connection, NMDAData *data, real *buffer, uinteger_t *firedTable, uinteger_t *firedTableSizes, size_t firedTableCap, size_t num, size_t offset, int time)
// __global__ void update_all_nmda_neuron(NMDAData *data, int num, int offset, int time)
{
	int currentIdx = time % (connection->maxDelay + 1);
	__shared__ uinteger_t fire_table_t[MAX_BLOCK_SIZE];
	__shared__ volatile uinteger_t fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}

	__syncthreads();

	size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (size_t idx = 0; idx < num; idx +=blockDim.x*gridDim.x) {
		bool fired = false;
		uinteger_t testLoc = 0;

		size_t nid = idx + tid;
		size_t gnid = offset + nid; 
		if (nid < num) {
			bool actived = data->pRefracStep[nid] <= 0;

			if (actived) {
				data->pV_m[nid] = data->pC_m[nid] * data->pV_m[nid] + data->pV_tmp[nid] + data->pI_e[nid] * data->pC_e[nid] + data->pI_i[nid] * data->pC_i[nid];

				// gXInput[gnid] += currentE[gnid] + currentI[gnid];

				data->pI_e[nid] *= data->pCe[nid];
				data->pI_i[nid] *= data->pCi[nid];

				fired = data->pV_m[nid] >= data->pV_thresh[nid];

				data->_fire_count[gnid] += fired;

				if (fired) {
					testLoc = atomicAdd((uinteger_t *)&fire_cnt, 1);
					if (testLoc < MAX_BLOCK_SIZE) {
						fire_table_t[testLoc] = gnid;
						fired = false;
					}

					data->pRefracStep[nid] = data->pRefracTime[nid] - 1;
					data->pV_m[nid] = data->pV_reset[nid];
				} else {
					data->pI_e[nid] += buffer[gnid];
					data->pI_i[nid] += buffer[gnid+num];
				}
			} else {
				data->pRefracStep[nid] = data->pRefracStep[nid] - 1;
			}

			buffer[gnid] = 0;
			buffer[gnid+num] = 0;
		}

		__syncthreads();
		if (fire_cnt >= MAX_BLOCK_SIZE) {
			commit2globalTable(fire_table_t, static_cast<uinteger_t>(MAX_BLOCK_SIZE), firedTable, &firedTableSizes[currentIdx], static_cast<uinteger_t>(firedTableCap*currentIdx));
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}

		__syncthreads();

		if (fired) {
			testLoc = atomicAdd((uinteger_t*)&fire_cnt, 1);
			if (testLoc < MAX_BLOCK_SIZE) {
				fire_table_t[testLoc] = gnid;
				fired = false;
			}
		}
		__syncthreads();
		if (fire_cnt >= MAX_BLOCK_SIZE) {
			commit2globalTable(fire_table_t, static_cast<uinteger_t>(MAX_BLOCK_SIZE), firedTable, &firedTableSizes[currentIdx], static_cast<uinteger_t>(firedTableCap*currentIdx));
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
		__syncthreads();

	}
	if (fire_cnt > 0) {
		commit2globalTable(fire_table_t, fire_cnt, firedTable, &firedTableSizes[currentIdx], static_cast<uinteger_t>(firedTableCap*currentIdx));
		if (threadIdx.x == 0) {
			fire_cnt = 0;
		}
	}
	__syncthreads();
}

// __global__ void update_dense_nmda_neuron(Connection *connection, NMDAData *data, real *buffer, int *firedTable, int *firedTableSizes, int firedTableCap, int num, int offset, int time)
// {
// 	//__shared__ int fire_table_t[MAX_BLOCK_SIZE];
// 	//__shared__ volatile int fire_cnt;
// 
// 	//if (threadIdx.x == 0) {
// 	//	fire_cnt = 0;
// 	//}
// 	//__syncthreads();
// 
// 	int tid = blockIdx.x * blockDim.x + threadIdx.x;
// 	int currentIdx = time % (connection->maxDelay+1);
// 	for (int idx = tid; idx < num; idx +=blockDim.x*gridDim.x) {
// 		//bool fired = false;
// 		//int testLoc = 0;
// 
// 		int nid = idx;
// 		int gnid = offset + idx; 
// 		bool actived = data->pRefracStep[idx] <= 0;
// 
// 		if (actived) {
// 			data->pV_m[nid] = data->pC_m[nid] * data->pV_m[nid] + data->pV_tmp[nid] + data->pI_e[nid] * data->pC_e[nid] + data->pI_i[nid] * data->pC_i[nid];
// 
// 			data->pI_e[nid] *= data->pCe[nid];
// 			data->pI_i[nid] *= data->pCi[nid];
// 
// 			bool fired = data->pV_m[nid] >= data->pV_thresh[nid];
// 
// 			firedTable[firedTableCap*currentIdx + gnid] = fired;
// 
// 			data->_fire_count[gnid] += fired;
// 
// 			if (fired) {
// 				data->pRefracStep[nid] = data->pRefracTime[nid] - 1;
// 				data->pV_m[nid] = data->pV_reset[nid];
// 
// 			} else {
// 				// gXInput[gnid] += currentE[gnid] + currentI[gnid];
// 				data->pI_e[nid] += buffer[gnid];
// 				data->pI_i[nid] += buffer[gnid+num];
// 				//real input = 0, input_I = 0;
// 				//for (int i=data->p_start_E[nid]; i<data->p_start_I[nid]; i++) {
// 				//	input += currentE[i];
// 				//}
// 				//for (int i=data->p_start_I[nid]; i<data->p_end[nid]; i++) {
// 				//	input_I += currentE[i];
// 				//}
// 				//data->pI_e[nid] += input;
// 				//data->pI_i[nid] += input_I;
// 				//gXInput[gnid] += input + input_I;
// 			}
// 
// 		} else {
// 			data->pRefracStep[idx] = data->pRefracStep[idx] - 1;
// 			firedTable[firedTableCap*currentIdx + gnid] = 0;
// 		}
// 		buffer[gnid] = 0;
// 		buffer[gnid+num] = 0;
// 	}
// 	__syncthreads();
// }

void cudaUpdateNMDA(Connection *conn, void *data, real *buffer, uinteger_t *firedTable, uinteger_t *firedTableSizes, size_t firedTableCap, size_t num, size_t offset, int time, BlockSize *pSize)
{
	// find_nmda_neuron<<<pSize->gridSize, pSize->blockSize>>>((NMDAData*)data, currentE, currentI, num, offset);
	// update_nmda_neuron<<<pSize->gridSize, pSize->blockSize>>>(conn, (NMDAData*)data, currentE, currentI, firedTable, firedTableSizes, num, offset, time);
	update_all_nmda_neuron<<<pSize->gridSize, pSize->blockSize>>>(conn, (NMDAData*)data, buffer, firedTable, firedTableSizes, firedTableCap, num, offset, time);

}
