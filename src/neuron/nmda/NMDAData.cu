#include <stdlib.h>
#include <string.h>
#include "../../../msg_utils/helper/helper_gpu.h"
#include "NMDAData.h"

void *cudaMallocNMDA()
{
	void *ret = NULL;
	checkCudaErrors(hipMalloc((void**)&(ret), sizeof(NMDAData)*1));
	checkCudaErrors(hipMemset(ret, 0, sizeof(NMDAData)*1));
	return ret;
}

void *cudaAllocNMDA(void *pCPU, size_t num)
{
	void *ret = cudaMallocNMDA();
	void *tmp = cudaAllocNMDAPara(pCPU, num);
	checkCudaErrors(hipMemcpy(ret, tmp, sizeof(NMDAData)*1, hipMemcpyHostToDevice));
	free(tmp);
	tmp = NULL;
	return ret;
}

void *cudaAllocNMDAPara(void *pCPU, size_t num)
{
	NMDAData *p = (NMDAData*)pCPU;
	NMDAData *ret = (NMDAData*)malloc(sizeof(NMDAData)*1);
	memset(ret, 0, sizeof(NMDAData)*1);

	checkCudaErrors(hipMalloc((void**)&(ret->pRefracTime), sizeof(int)*num));
	checkCudaErrors(hipMemset(ret->pRefracTime, 0, sizeof(int)*num));
	checkCudaErrors(hipMemcpy(ret->pRefracTime, p->pRefracTime, sizeof(int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pRefracStep), sizeof(int)*num));
	checkCudaErrors(hipMemset(ret->pRefracStep, 0, sizeof(int)*num));
	checkCudaErrors(hipMemcpy(ret->pRefracStep, p->pRefracStep, sizeof(int)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(ret->pI_e), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pI_e, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pI_e, p->pI_e, sizeof(real)*num, hipMemcpyHostToDevice));
	// checkCudaErrors(hipMalloc((void**)&(ret->pV_i), sizeof(real)*num));
	// checkCudaErrors(hipMemset(ret->pV_i, 0, sizeof(real)*num));
	// checkCudaErrors(hipMemcpy(ret->pV_i, p->pV_i, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pCe), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pCe, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pCe, p->pCe, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pV_reset), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pV_reset, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pV_reset, p->pV_reset, sizeof(real)*num, hipMemcpyHostToDevice));
	// checkCudaErrors(hipMalloc((void**)&(ret->pV_e), sizeof(real)*num));
	// checkCudaErrors(hipMemset(ret->pV_e, 0, sizeof(real)*num));
	// checkCudaErrors(hipMemcpy(ret->pV_e, p->pV_e, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pV_tmp), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pV_tmp, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pV_tmp, p->pV_tmp, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pI_i), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pI_i, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pI_i, p->pI_i, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pV_thresh), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pV_thresh, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pV_thresh, p->pV_thresh, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pCi), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pCi, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pCi, p->pCi, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pV_m), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pV_m, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pV_m, p->pV_m, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pC_e), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pC_e, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pC_e, p->pC_e, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pC_m), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pC_m, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pC_m, p->pC_m, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pC_i), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pC_i, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pC_i, p->pC_i, sizeof(real)*num, hipMemcpyHostToDevice));

	ret->_fire_count = TOGPU(p->_fire_count, num);

	return ret;
}

int cudaFetchNMDA(void *pCPU, void *pGPU, size_t num)
{
	NMDAData *pTmp = (NMDAData*)malloc(sizeof(NMDAData)*1);
	memset(pTmp, 0, sizeof(NMDAData)*1);
	checkCudaErrors(hipMemcpy(pTmp, pGPU, sizeof(NMDAData)*1, hipMemcpyDeviceToHost));

	cudaNMDAParaFromGPU(pCPU, pTmp, num);
	return 0;
}

int cudaNMDAParaToGPU(void *pCPU, void *pGPU, size_t num)
{
	NMDAData *pC = (NMDAData*)pCPU;
	NMDAData *pG = (NMDAData*)pGPU;

	checkCudaErrors(hipMemcpy(pG->pRefracTime, pC->pRefracTime, sizeof(int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pRefracStep, pC->pRefracStep, sizeof(int)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(pG->pI_e, pC->pI_e, sizeof(real)*num, hipMemcpyHostToDevice));
	// checkCudaErrors(hipMemcpy(pG->pV_i, pC->pV_i, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pCe, pC->pCe, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pV_reset, pC->pV_reset, sizeof(real)*num, hipMemcpyHostToDevice));
	// checkCudaErrors(hipMemcpy(pG->pV_e, pC->pV_e, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pV_tmp, pC->pV_tmp, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pI_i, pC->pI_i, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pV_thresh, pC->pV_thresh, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pCi, pC->pCi, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pV_m, pC->pV_m, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pC_e, pC->pC_e, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pC_m, pC->pC_m, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pC_i, pC->pC_i, sizeof(real)*num, hipMemcpyHostToDevice));

	COPYTOGPU(pG->_fire_count, pC->_fire_count, num);

	return 0;
}

int cudaNMDAParaFromGPU(void *pCPU, void *pGPU, size_t num)
{
	NMDAData *pC = (NMDAData*)pCPU;
	NMDAData *pG = (NMDAData*)pGPU;

	checkCudaErrors(hipMemcpy(pC->pRefracTime, pG->pRefracTime, sizeof(int)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pRefracStep, pG->pRefracStep, sizeof(int)*num, hipMemcpyDeviceToHost));

	checkCudaErrors(hipMemcpy(pC->pI_e, pG->pI_e, sizeof(real)*num, hipMemcpyDeviceToHost));
	// checkCudaErrors(hipMemcpy(pC->pV_i, pG->pV_i, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pCe, pG->pCe, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pV_reset, pG->pV_reset, sizeof(real)*num, hipMemcpyDeviceToHost));
	// checkCudaErrors(hipMemcpy(pC->pV_e, pG->pV_e, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pV_tmp, pG->pV_tmp, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pI_i, pG->pI_i, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pV_thresh, pG->pV_thresh, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pCi, pG->pCi, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pV_m, pG->pV_m, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pC_e, pG->pC_e, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pC_m, pG->pC_m, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pC_i, pG->pC_i, sizeof(real)*num, hipMemcpyDeviceToHost));

	COPYFROMGPU(pC->_fire_count, pG->_fire_count, num);

	return 0;
}

int cudaFreeNMDA(void *pGPU)
{
	NMDAData *tmp = (NMDAData*)malloc(sizeof(NMDAData)*1);
	memset(tmp, 0, sizeof(NMDAData)*1);
	checkCudaErrors(hipMemcpy(tmp, pGPU, sizeof(NMDAData)*1, hipMemcpyDeviceToHost));
	cudaFreeNMDAPara(tmp);
	free(tmp);
	tmp = NULL;
	hipFree(pGPU);
	pGPU = NULL;
	return 0;
}

int cudaFreeNMDAPara(void *pGPU)
{
	NMDAData *p = (NMDAData*)pGPU;
	hipFree(p->pRefracTime);
	p->pRefracTime = NULL;
	hipFree(p->pRefracStep);
	p->pRefracStep = NULL;

	hipFree(p->pI_e);
	p->pI_e = NULL;
	// hipFree(p->pV_i);
	// p->pV_i = NULL;
	hipFree(p->pCe);
	p->pCe = NULL;
	hipFree(p->pV_reset);
	p->pV_reset = NULL;
	// hipFree(p->pV_e);
	// p->pV_e = NULL;
	hipFree(p->pV_tmp);
	p->pV_tmp = NULL;
	hipFree(p->pI_i);
	p->pI_i = NULL;
	hipFree(p->pV_thresh);
	p->pV_thresh = NULL;
	hipFree(p->pCi);
	p->pCi = NULL;
	hipFree(p->pV_m);
	p->pV_m = NULL;
	hipFree(p->pC_e);
	p->pC_e = NULL;
	hipFree(p->pC_m);
	p->pC_m = NULL;
	hipFree(p->pC_i);
	p->pC_i = NULL;

	gpuFree(p->_fire_count);

	return 0;
}

int cudaLogRateNMDA(void *cpu, void *gpu, const char *name)
{
	NMDAData *c = static_cast<NMDAData *>(cpu);
	NMDAData *g = static_cast<NMDAData *>(gpu);

	NMDAData *t = FROMGPU(g, 1);
	COPYFROMGPU(c->_fire_count, t->_fire_count, c->num);
	return logRateNMDA(cpu, name);
}

real * cudaGetVNMDA(void *data) {
	NMDAData *c_g_nmda = FROMGPU(static_cast<NMDAData *>(data), 1);
	return c_g_nmda->pV_m;
}
