#include <stdlib.h>
#include <string.h>
#include "../../../msg_utils/helper/helper_gpu.h"
#include "../../../msg_utils/helper/helper_c.h"
#include "LIFExpData.h"

void *cudaMallocLIFExp()
{
	void *ret = NULL;
	checkCudaErrors(hipMalloc((void**)&(ret), sizeof(LIFExpData)*1));
	checkCudaErrors(hipMemset(ret, 0, sizeof(LIFExpData)*1));
	return ret;
}

void *cudaAllocLIFExp(void *pCPU, size_t num)
{
	void *ret = cudaMallocLIFExp();
	void *tmp = cudaAllocLIFExpPara(pCPU, num);
	checkCudaErrors(hipMemcpy(ret, tmp, sizeof(LIFExpData)*1, hipMemcpyHostToDevice));
	free(tmp);
	tmp = NULL;
	return ret;
}

void *cudaAllocLIFExpPara(void *pCPU, size_t num)
{
	LIFExpData *p = (LIFExpData*)pCPU;
	LIFExpData *ret = (LIFExpData*)malloc(sizeof(LIFExpData)*1);
	memset(ret, 0, sizeof(LIFExpData)*1);

	checkCudaErrors(hipMalloc((void**)&(ret->pRefracTime), sizeof(int)*num));
	checkCudaErrors(hipMemset(ret->pRefracTime, 0, sizeof(int)*num));
	checkCudaErrors(hipMemcpy(ret->pRefracTime, p->pRefracTime, sizeof(int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pRefracStep), sizeof(int)*num));
	checkCudaErrors(hipMemset(ret->pRefracStep, 0, sizeof(int)*num));
	checkCudaErrors(hipMemcpy(ret->pRefracStep, p->pRefracStep, sizeof(int)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(ret->pV), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pV, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pV, p->pV, sizeof(real)*num, hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void**)&(ret->pV_tmp), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pV_tmp, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pV_tmp, p->pV_tmp, sizeof(real)*num, hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void**)&(ret->pV_thresh), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pV_thresh, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pV_thresh, p->pV_thresh, sizeof(real)*num, hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void**)&(ret->pV_reset), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pV_reset, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pV_reset, p->pV_reset, sizeof(real)*num, hipMemcpyHostToDevice));

    
    checkCudaErrors(hipMalloc((void**)&(ret->pR), sizeof(real)*num));
    checkCudaErrors(hipMemset(ret->pR, 0, sizeof(real)*num));
    checkCudaErrors(hipMemcpy(ret->pR, p->pR, sizeof(real)*num, hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void**)&(ret->pC_m), sizeof(real)*num));
    checkCudaErrors(hipMemset(ret->pC_m, 0, sizeof(real)*num));
    checkCudaErrors(hipMemcpy(ret->pC_m, p->pC_m, sizeof(real)*num, hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void**)&(ret->pE), sizeof(real)*num));
    checkCudaErrors(hipMemset(ret->pE, 0, sizeof(real)*num));
    checkCudaErrors(hipMemcpy(ret->pE, p->pE, sizeof(real)*num, hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void**)&(ret->pI), sizeof(real)*num));
    checkCudaErrors(hipMemset(ret->pI, 0, sizeof(real)*num));
    checkCudaErrors(hipMemcpy(ret->pI, p->pI, sizeof(real)*num, hipMemcpyHostToDevice));

	ret->_fire_count = TOGPU(p->_fire_count, num);

	return ret;
}

int cudaFetchLIFExp(void *pCPU, void *pGPU, size_t num)
{
	LIFExpData *pTmp = (LIFExpData*)malloc(sizeof(LIFExpData)*1);
	memset(pTmp, 0, sizeof(LIFExpData)*1);
	checkCudaErrors(hipMemcpy(pTmp, pGPU, sizeof(LIFExpData)*1, hipMemcpyDeviceToHost));

	cudaLIFExpParaFromGPU(pCPU, pTmp, num);
	return 0;
}

int cudaLIFExpParaToGPU(void *pCPU, void *pGPU, size_t num)
{
	LIFExpData *pC = (LIFExpData*)pCPU;
	LIFExpData *pG = (LIFExpData*)pGPU;

	checkCudaErrors(hipMemcpy(pG->pRefracTime, pC->pRefracTime, sizeof(int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pRefracStep, pC->pRefracStep, sizeof(int)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(pG->pV, pC->pV, sizeof(real)*num, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(pG->pV_tmp, pC->pV_tmp, sizeof(real)*num, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(pG->pV_thresh, pC->pV_thresh, sizeof(real)*num, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(pG->pV_reset, pC->pV_reset, sizeof(real)*num, hipMemcpyHostToDevice));

    checkCudaErrors(hipMemcpy(pG->pR, pC->pR, sizeof(real)*num, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(pG->pC_m, pC->pC_m, sizeof(real)*num, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(pG->pE, pC->pE, sizeof(real)*num, hipMemcpyHostToDevice));

    checkCudaErrors(hipMemcpy(pG->pI, pC->pI, sizeof(real)*num, hipMemcpyHostToDevice));

	COPYTOGPU(pG->_fire_count, pC->_fire_count, num);

	return 0;
}

int cudaLIFExpParaFromGPU(void *pCPU, void *pGPU, size_t num)
{
	LIFExpData *pC = (LIFExpData*)pCPU;
	LIFExpData *pG = (LIFExpData*)pGPU;

	checkCudaErrors(hipMemcpy(pC->pRefracTime, pG->pRefracTime, sizeof(int)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pRefracStep, pG->pRefracStep, sizeof(int)*num, hipMemcpyDeviceToHost));

	checkCudaErrors(hipMemcpy(pC->pV, pG->pV, sizeof(real)*num, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(pC->pV_tmp, pG->pV_tmp, sizeof(real)*num, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(pC->pV_thresh, pG->pV_thresh, sizeof(real)*num, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(pC->pV_reset, pG->pV_reset, sizeof(real)*num, hipMemcpyDeviceToHost));

    checkCudaErrors(hipMemcpy(pC->pR, pG->pR, sizeof(real)*num, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(pC->pC_m, pG->pC_m, sizeof(real)*num, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(pC->pE, pG->pE, sizeof(real)*num, hipMemcpyDeviceToHost));

    checkCudaErrors(hipMemcpy(pC->pI, pG->pI, sizeof(real)*num, hipMemcpyDeviceToHost));

	COPYFROMGPU(pC->_fire_count, pG->_fire_count, num);

	return 0;
}

int cudaFreeLIFExp(void *pGPU)
{
	LIFExpData *tmp = (LIFExpData*)malloc(sizeof(LIFExpData)*1);
	memset(tmp, 0, sizeof(LIFExpData)*1);
	checkCudaErrors(hipMemcpy(tmp, pGPU, sizeof(LIFExpData)*1, hipMemcpyDeviceToHost));
	cudaFreeLIFExpPara(tmp);
	tmp = free_c(tmp);
	hipFree(pGPU);
	pGPU = NULL;
	return 0;
}

int cudaFreeLIFExpPara(void *pGPU)
{
	LIFExpData *p = (LIFExpData*)pGPU;
	hipFree(p->pRefracTime);
	p->pRefracTime = NULL;
	hipFree(p->pRefracStep);
	p->pRefracStep = NULL;

	hipFree(p->pV);
	p->pV = NULL;
    hipFree(p->pV_tmp);
    p->pV_tmp = NULL;
    hipFree(p->pV_thresh);
    p->pV_thresh = NULL;
    hipFree(p->pV_reset);
    p->pV_reset = NULL;

    hipFree(p->pR);
    p->pR = NULL;
    hipFree(p->pC_m);
    p->pC_m = NULL;
    hipFree(p->pE);
    p->pE = NULL;

    hipFree(p->pI);
    p->pI = NULL;

	gpuFree(p->_fire_count);

	return 0;
}

int cudaLogRateLIFExp(void *cpu, void *gpu, const char *name)
{
	LIFExpData *c = static_cast<LIFExpData *>(cpu);
	LIFExpData *g = static_cast<LIFExpData *>(gpu);

	LIFExpData *t = FROMGPU(g, 1);
	COPYFROMGPU(c->_fire_count, t->_fire_count, c->num);
	return logRateLIFExp(cpu, name);
}

real * cudaGetVLIFExp(void *data) {
	LIFExpData *c_g_lif = FROMGPU(static_cast<LIFExpData *>(data), 1);
	return c_g_lif->pV;
}
