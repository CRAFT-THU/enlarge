#include "hip/hip_runtime.h"

#include "LIFData.h"

#include "../../../msg_utils/helper/helper_gpu.h"
#include "../../gpu_utils/runtime.h"
#include "../../net/Connection.h"

// __global__ void find_lif_neuron(LIFData *data, real * currentE, real * currentI, int num, int offset)
// {
// 	__shared__ uinteger_t tActiveTable[MAX_BLOCK_SIZE];
// 	__shared__ volatile uinteger_t activeCnt;
// 
// 	if (threadIdx.x == 0) {
// 		activeCnt = 0;
// 	}
// 	__syncthreads();
// 
// 	uinteger_t tid = blockIdx.x * blockDim.x + threadIdx.x;
// 	for (uinteger_t idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
// 		//bool actived = false;
// 		int testLoc = 0;
// 		bool actived = data->pRefracStep[idx] <= 0;
// 		if (actived) {
// 			testLoc = atomicAdd((int*)&activeCnt, 1);
// 			if (testLoc < MAX_BLOCK_SIZE) {
// 				tActiveTable[testLoc] = idx;
// 				actived = false;
// 			}
// 		} else {
// 			currentE[offset + idx] = 0;
// 			currentI[offset + idx] = 0;
// 			data->pRefracStep[idx] = data->pRefracStep[idx] - 1;
// 		}
// 		__syncthreads();
// 
// 		if (activeCnt >= MAX_BLOCK_SIZE) {
// 			commit2globalTable(tActiveTable, MAX_BLOCK_SIZE, gActiveTable, &gActiveTableSize, 0);
// 			if (threadIdx.x == 0) {
// 				activeCnt = 0;
// 			}
// 		}
// 		__syncthreads();
// 
// 		if (actived) {
// 			testLoc = atomicAdd((int*)&activeCnt, 1);
// 			if (testLoc < MAX_BLOCK_SIZE) {
// 				tActiveTable[testLoc] = idx;
// 				actived = false;
// 			}
// 		}
// 		__syncthreads();
// 
// 		if (activeCnt >= MAX_BLOCK_SIZE) {
// 			commit2globalTable(tActiveTable, MAX_BLOCK_SIZE, gActiveTable, &gActiveTableSize, 0);
// 			if (threadIdx.x == 0) {
// 				activeCnt = 0;
// 			}
// 		}
// 		__syncthreads();
// 
// 		if (activeCnt > 0) {
// 			commit2globalTable(tActiveTable, activeCnt, gActiveTable, &gActiveTableSize, 0);
// 			if (threadIdx.x == 0) {
// 				activeCnt = 0;
// 			}
// 		}
// 		__syncthreads();
// 	}
// }

// __global__ void update_lif_neuron(Connection *connection, LIFData *data, real *currentE, real *currentI, int *firedTable, int *firedTableSizes, int num, int offset, int time)
// {
// 	int currentIdx = time % (connection->maxDelay+1);
// 	__shared__ int fire_table_t[MAX_BLOCK_SIZE];
// 	__shared__ volatile int fire_cnt;
// 	if (threadIdx.x == 0) {
// 		fire_cnt = 0;
// 	}
// 	__syncthreads();
// 
// 	int tid = blockIdx.x * blockDim.x + threadIdx.x;
// 	for (int idx = tid; idx < gActiveTableSize; idx +=blockDim.x*gridDim.x) {
// 		bool fired = false;
// 		int testLoc = 0;
// 
// 		int nid = gActiveTable[idx];
// 		int gnid = offset + nid; 
// 
// 		//real I = sqrtf(data->pCe[nid]) * data->pI_e[nid] + sqrtf(data->pCi[nid]) * data->pI_i[nid] + data->p_i_tmp[nid];
// 
// 		//real I = currentE[gnid] + data->p_i_tmp[nid];
// 		//data->pV_m[nid] = data->pV_m[nid] * data->p_C1[nid] + data->p_C2[nid] * I;
// 
// 		data->pV_m[nid] = data->pC_m[nid] * data->pV_m[nid] + data->pV_tmp[nid] + data->pI_e[nid] * data->pC_e[nid] + data->pI_i[nid] * data->pC_i[nid];
// 
// 		//data->p_i_syn[nid] = 0;
// 
// 		data->pI_e[nid] *= data->pCe[nid];
// 		data->pI_i[nid] *= data->pCi[nid];
// 
// 		fired = data->pV_m[nid] >= data->pV_thresh[nid];
// 
// 		gFireCount[gnid] += fired;
// 
// 		if (fired) {
// 			testLoc = atomicAdd((int*)&fire_cnt, 1);
// 			if (testLoc < MAX_BLOCK_SIZE) {
// 				fire_table_t[testLoc] = gnid;
// 				fired = false;
// 			}
// 
// 			data->pRefracStep[nid] = data->pRefracTime[nid] - 1;
// 			data->pV_m[nid] = data->pV_reset[nid];
// 		} else {
// 			gXInput[gnid] += currentE[gnid] + currentI[gnid];
// 			data->pI_e[nid] += currentE[gnid];
// 			data->pI_i[nid] += currentI[gnid];
// 		}
// 
// 		currentE[gnid] = 0;
// 		currentI[gnid] = 0;
// 
// 		__syncthreads();
// 		if (fire_cnt >= MAX_BLOCK_SIZE) {
// 			commit2globalTable(fire_table_t, MAX_BLOCK_SIZE, firedTable, &firedTableSizes[currentIdx], gFiredTableCap*currentIdx);
// 			if (threadIdx.x == 0) {
// 				fire_cnt = 0;
// 			}
// 		}
// 
// 		__syncthreads();
// 
// 		if (fired) {
// 			testLoc = atomicAdd((int*)&fire_cnt, 1);
// 			if (testLoc < MAX_BLOCK_SIZE) {
// 				fire_table_t[testLoc] = gnid;
// 				fired = false;
// 			}
// 		}
// 		__syncthreads();
// 		if (fire_cnt >= MAX_BLOCK_SIZE) {
// 			commit2globalTable(fire_table_t, MAX_BLOCK_SIZE, firedTable, &firedTableSizes[currentIdx], gFiredTableCap*currentIdx);
// 			if (threadIdx.x == 0) {
// 				fire_cnt = 0;
// 			}
// 		}
// 		__syncthreads();
// 
// 		if (fire_cnt > 0) {
// 			commit2globalTable(fire_table_t, fire_cnt, firedTable, &firedTableSizes[currentIdx], gFiredTableCap*currentIdx);
// 			if (threadIdx.x == 0) {
// 				fire_cnt = 0;
// 			}
// 		}
// 
// 	}
// 	//__syncthreads();
// 	//if (threadIdx.x == 0 && blockIdx.x == 0) {
// 	//	gActiveTableSize = 0;
// 	//}
// }

__global__ void update_all_lif_neuron(Connection *connection, LIFData *data, real *buffer, uinteger_t *firedTable, uinteger_t *firedTableSizes, size_t firedTableCap, size_t num, size_t offset, int time)
// __global__ void update_all_lif_neuron(LIFData *data, int num, int offset, int time)
{
	int currentIdx = time % (connection->maxDelay + 1);
	__shared__ uinteger_t fire_table_t[MAX_BLOCK_SIZE];
	__shared__ volatile uinteger_t fire_cnt;

	if (threadIdx.x == 0) {
		fire_cnt = 0;
	}

	__syncthreads();

	size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (size_t idx = 0; idx < num; idx +=blockDim.x*gridDim.x) {
		bool fired = false;
		uinteger_t testLoc = 0;

		size_t nid = idx + tid;
		size_t gnid = offset + nid; 
		if (nid < num) {
			bool actived = data->pRefracStep[nid] <= 0;

			if (actived) {
				data->pV_m[nid] = data->pC_m[nid] * data->pV_m[nid] + data->pV_tmp[nid] + data->pI_e[nid] * data->pC_e[nid] + data->pI_i[nid] * data->pC_i[nid];

				// gXInput[gnid] += currentE[gnid] + currentI[gnid];

				data->pI_e[nid] *= data->pCe[nid];
				data->pI_i[nid] *= data->pCi[nid];

				fired = data->pV_m[nid] >= data->pV_thresh[nid];

				data->_fire_count[gnid] += fired;

				if (fired) {
					testLoc = atomicAdd((uinteger_t *)&fire_cnt, 1);
					if (testLoc < MAX_BLOCK_SIZE) {
						fire_table_t[testLoc] = gnid;
						fired = false;
					}

					data->pRefracStep[nid] = data->pRefracTime[nid] - 1;
					data->pV_m[nid] = data->pV_reset[nid];
				} else {
					int input_start = data->pInput_start[nid];
					int input_end = data->pInput_start[nid+1];
					int input_idx = input_start + time;

					if (input_start == input_end) { // 不使用初始输入
						data->pI_e[nid] += buffer[gnid];
						data->pI_i[nid] += buffer[gnid+num];
					} else if (input_idx < input_end) {
						data->pI_e[nid] += data->pInput[input_idx];
						data->pI_i[nid] += data->pInput[input_idx]; // FIXME: are they the same?
					} // else do nth
				}
			} else {
				data->pRefracStep[nid] = data->pRefracStep[nid] - 1;
			}

			buffer[gnid] = 0;
			buffer[gnid+num] = 0;
		}

		__syncthreads();
		if (fire_cnt >= MAX_BLOCK_SIZE) {
			commit2globalTable(fire_table_t, static_cast<uinteger_t>(MAX_BLOCK_SIZE), firedTable, &firedTableSizes[currentIdx], static_cast<uinteger_t>(firedTableCap*currentIdx));
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}

		__syncthreads();

		if (fired) {
			testLoc = atomicAdd((uinteger_t*)&fire_cnt, 1);
			if (testLoc < MAX_BLOCK_SIZE) {
				fire_table_t[testLoc] = gnid;
				fired = false;
			}
		}
		__syncthreads();
		if (fire_cnt >= MAX_BLOCK_SIZE) {
			commit2globalTable(fire_table_t, static_cast<uinteger_t>(MAX_BLOCK_SIZE), firedTable, &firedTableSizes[currentIdx], static_cast<uinteger_t>(firedTableCap*currentIdx));
			if (threadIdx.x == 0) {
				fire_cnt = 0;
			}
		}
		__syncthreads();

	}
	if (fire_cnt > 0) {
		commit2globalTable(fire_table_t, fire_cnt, firedTable, &firedTableSizes[currentIdx], static_cast<uinteger_t>(firedTableCap*currentIdx));
		if (threadIdx.x == 0) {
			fire_cnt = 0;
		}
	}
	__syncthreads();
}

// __global__ void update_dense_lif_neuron(Connection *connection, LIFData *data, real *buffer, int *firedTable, int *firedTableSizes, int firedTableCap, int num, int offset, int time)
// {
// 	//__shared__ int fire_table_t[MAX_BLOCK_SIZE];
// 	//__shared__ volatile int fire_cnt;
// 
// 	//if (threadIdx.x == 0) {
// 	//	fire_cnt = 0;
// 	//}
// 	//__syncthreads();
// 
// 	int tid = blockIdx.x * blockDim.x + threadIdx.x;
// 	int currentIdx = time % (connection->maxDelay+1);
// 	for (int idx = tid; idx < num; idx +=blockDim.x*gridDim.x) {
// 		//bool fired = false;
// 		//int testLoc = 0;
// 
// 		int nid = idx;
// 		int gnid = offset + idx; 
// 		bool actived = data->pRefracStep[idx] <= 0;
// 
// 		if (actived) {
// 			data->pV_m[nid] = data->pC_m[nid] * data->pV_m[nid] + data->pV_tmp[nid] + data->pI_e[nid] * data->pC_e[nid] + data->pI_i[nid] * data->pC_i[nid];
// 
// 			data->pI_e[nid] *= data->pCe[nid];
// 			data->pI_i[nid] *= data->pCi[nid];
// 
// 			bool fired = data->pV_m[nid] >= data->pV_thresh[nid];
// 
// 			firedTable[firedTableCap*currentIdx + gnid] = fired;
// 
// 			data->_fire_count[gnid] += fired;
// 
// 			if (fired) {
// 				data->pRefracStep[nid] = data->pRefracTime[nid] - 1;
// 				data->pV_m[nid] = data->pV_reset[nid];
// 
// 			} else {
// 				// gXInput[gnid] += currentE[gnid] + currentI[gnid];
// 				data->pI_e[nid] += buffer[gnid];
// 				data->pI_i[nid] += buffer[gnid+num];
// 				//real input = 0, input_I = 0;
// 				//for (int i=data->p_start_E[nid]; i<data->p_start_I[nid]; i++) {
// 				//	input += currentE[i];
// 				//}
// 				//for (int i=data->p_start_I[nid]; i<data->p_end[nid]; i++) {
// 				//	input_I += currentE[i];
// 				//}
// 				//data->pI_e[nid] += input;
// 				//data->pI_i[nid] += input_I;
// 				//gXInput[gnid] += input + input_I;
// 			}
// 
// 		} else {
// 			data->pRefracStep[idx] = data->pRefracStep[idx] - 1;
// 			firedTable[firedTableCap*currentIdx + gnid] = 0;
// 		}
// 		buffer[gnid] = 0;
// 		buffer[gnid+num] = 0;
// 	}
// 	__syncthreads();
// }

void cudaUpdateLIF(Connection *conn, void *data, real *buffer, uinteger_t *firedTable, uinteger_t *firedTableSizes, size_t firedTableCap, size_t num, size_t offset, int time, BlockSize *pSize)
{
	// find_lif_neuron<<<pSize->gridSize, pSize->blockSize>>>((LIFData*)data, currentE, currentI, num, offset);
	// update_lif_neuron<<<pSize->gridSize, pSize->blockSize>>>(conn, (LIFData*)data, currentE, currentI, firedTable, firedTableSizes, num, offset, time);
	update_all_lif_neuron<<<pSize->gridSize, pSize->blockSize>>>(conn, (LIFData*)data, buffer, firedTable, firedTableSizes, firedTableCap, num, offset, time);

}
