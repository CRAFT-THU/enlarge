#include <stdlib.h>
#include <string.h>
#include "../../../msg_utils/helper/helper_gpu.h"
#include "IAFData.h"

void *cudaMallocIAF()
{
	void *ret = NULL;
	checkCudaErrors(hipMalloc((void**)&(ret), sizeof(IAFData)*1));
	checkCudaErrors(hipMemset(ret, 0, sizeof(IAFData)*1));
	return ret;
}

void *cudaAllocIAF(void *pCPU, size_t num)
{
	void *ret = cudaMallocIAF();
	void *tmp = cudaAllocIAFPara(pCPU, num);
	checkCudaErrors(hipMemcpy(ret, tmp, sizeof(IAFData)*1, hipMemcpyHostToDevice));
	free(tmp);
	tmp = NULL;
	return ret;
}

void *cudaAllocIAFPara(void *pCPU, size_t num)
{
	IAFData *p = (IAFData*)pCPU;
	IAFData *ret = (IAFData*)malloc(sizeof(IAFData)*1);
	memset(ret, 0, sizeof(IAFData)*1);

	checkCudaErrors(hipMalloc((void**)&(ret->pRefracTime), sizeof(int)*num));
	checkCudaErrors(hipMemset(ret->pRefracTime, 0, sizeof(int)*num));
	checkCudaErrors(hipMemcpy(ret->pRefracTime, p->pRefracTime, sizeof(int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(ret->pRefracStep), sizeof(int)*num));
	checkCudaErrors(hipMemset(ret->pRefracStep, 0, sizeof(int)*num));
	checkCudaErrors(hipMemcpy(ret->pRefracStep, p->pRefracStep, sizeof(int)*num, hipMemcpyHostToDevice));

	// model param
	checkCudaErrors(hipMalloc((void**)&(ret->pE_L), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pE_L, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pE_L, p->pE_L, sizeof(real)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(ret->pI_e), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pI_e, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pI_e, p->pI_e, sizeof(real)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(ret->pTheta), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pTheta, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pTheta, p->pTheta, sizeof(real)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(ret->pV_reset), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pV_reset, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pV_reset, p->pV_reset, sizeof(real)*num, hipMemcpyHostToDevice));

	// state param
	checkCudaErrors(hipMalloc((void**)&(ret->pi_0), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pi_0, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pi_0, p->pi_0, sizeof(real)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(ret->pi_1), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pi_1, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pi_1, p->pi_1, sizeof(real)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(ret->pi_syn_ex), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pi_syn_ex, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pi_syn_ex, p->pi_syn_ex, sizeof(real)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(ret->pi_syn_in), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pi_syn_in, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pi_syn_in, p->pi_syn_in, sizeof(real)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(ret->pV_m), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pV_m, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pV_m, p->pV_m, sizeof(real)*num, hipMemcpyHostToDevice));

	// internal param
	checkCudaErrors(hipMalloc((void**)&(ret->pP11ex), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pP11ex, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pP11ex, p->pP11ex, sizeof(real)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(ret->pP11in), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pP11in, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pP11in, p->pP11in, sizeof(real)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(ret->pP22), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pP22, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pP22, p->pP22, sizeof(real)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(ret->pP21ex), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pP21ex, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pP21ex, p->pP21ex, sizeof(real)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(ret->pP21in), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pP21in, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pP21in, p->pP21in, sizeof(real)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(ret->pP20), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pP20, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pP20, p->pP20, sizeof(real)*num, hipMemcpyHostToDevice));


	ret->_fire_count = TOGPU(p->_fire_count, num);

	return ret;
}

int cudaFetchIAF(void *pCPU, void *pGPU, size_t num)
{
	IAFData *pTmp = (IAFData*)malloc(sizeof(IAFData)*1);
	memset(pTmp, 0, sizeof(IAFData)*1);
	checkCudaErrors(hipMemcpy(pTmp, pGPU, sizeof(IAFData)*1, hipMemcpyDeviceToHost));

	cudaIAFParaFromGPU(pCPU, pTmp, num);
	return 0;
}

int cudaIAFParaToGPU(void *pCPU, void *pGPU, size_t num)
{
	IAFData *pC = (IAFData*)pCPU;
	IAFData *pG = (IAFData*)pGPU;

	checkCudaErrors(hipMemcpy(pG->pRefracTime, pC->pRefracTime, sizeof(int)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pRefracStep, pC->pRefracStep, sizeof(int)*num, hipMemcpyHostToDevice));

	// model param
	checkCudaErrors(hipMemcpy(pG->pE_L, pC->pE_L, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pI_e, pC->pI_e, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pTheta, pC->pTheta, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pV_reset, pC->pV_reset, sizeof(real)*num, hipMemcpyHostToDevice));

	// state param
	checkCudaErrors(hipMemcpy(pG->pi_0, pC->pi_0, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pi_1, pC->pi_1, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pi_syn_ex, pC->pi_syn_ex, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pi_syn_in, pC->pi_syn_in, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pV_m, pC->pV_m, sizeof(real)*num, hipMemcpyHostToDevice));

	// internal param
	checkCudaErrors(hipMemcpy(pG->pP11ex, pC->pP11ex, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pP11in, pC->pP11in, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pP22, pC->pP22, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pP21ex, pC->pP21ex, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pP21in, pC->pP21in, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pP20, pC->pP20, sizeof(real)*num, hipMemcpyHostToDevice));

	COPYTOGPU(pG->_fire_count, pC->_fire_count, num);

	return 0;
}

int cudaIAFParaFromGPU(void *pCPU, void *pGPU, size_t num)
{
	IAFData *pC = (IAFData*)pCPU;
	IAFData *pG = (IAFData*)pGPU;

	checkCudaErrors(hipMemcpy(pC->pRefracTime, pG->pRefracTime, sizeof(int)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pRefracStep, pG->pRefracStep, sizeof(int)*num, hipMemcpyDeviceToHost));

	checkCudaErrors(hipMemcpy(pC->pE_L, pG->pE_L, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pI_e, pG->pI_e, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pTheta, pG->pTheta, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pV_reset, pG->pV_reset, sizeof(real)*num, hipMemcpyDeviceToHost));

	checkCudaErrors(hipMemcpy(pC->pi_0, pG->pi_0, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pi_1, pG->pi_1, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pi_syn_ex, pG->pi_syn_ex, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pi_syn_in, pG->pi_syn_in, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pV_m, pG->pV_m, sizeof(real)*num, hipMemcpyDeviceToHost));
	
	checkCudaErrors(hipMemcpy(pC->pP11ex, pG->pP11ex, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pP11in, pG->pP11in, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pP22, pG->pP22, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pP21ex, pG->pP21ex, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pP21in, pG->pP21in, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pP20, pG->pP20, sizeof(real)*num, hipMemcpyDeviceToHost));

	COPYFROMGPU(pC->_fire_count, pG->_fire_count, num);

	return 0;
}

int cudaFreeIAF(void *pGPU)
{
	IAFData *tmp = (IAFData*)malloc(sizeof(IAFData)*1);
	memset(tmp, 0, sizeof(IAFData)*1);
	checkCudaErrors(hipMemcpy(tmp, pGPU, sizeof(IAFData)*1, hipMemcpyDeviceToHost));
	cudaFreeIAFPara(tmp);
	free(tmp);
	tmp = NULL;
	hipFree(pGPU);
	pGPU = NULL;
	return 0;
}

int cudaFreeIAFPara(void *pGPU)
{
	IAFData *p = (IAFData*)pGPU;
	hipFree(p->pRefracTime);
	p->pRefracTime = NULL;
	hipFree(p->pRefracStep);
	p->pRefracStep = NULL;

	hipFree(p->pE_L);
	p->pE_L = NULL;
	hipFree(p->pI_e);
	p->pI_e = NULL;
	hipFree(p->pTheta);
	p->pTheta = NULL;
	hipFree(p->pV_reset);
	p->pV_reset = NULL;

	hipFree(p->pi_0);
	p->pi_0 = NULL;
	hipFree(p->pi_1);
	p->pi_1 = NULL;
	hipFree(p->pi_syn_ex);
	p->pi_syn_ex = NULL;
	hipFree(p->pi_syn_in);
	p->pi_syn_in = NULL;
	hipFree(p->pV_m);
	p->pV_m = NULL;

	hipFree(p->pP11ex);
	p->pP11ex = NULL;
	hipFree(p->pP11in);
	p->pP11in = NULL;
	hipFree(p->pP22);
	p->pP22 = NULL;
	hipFree(p->pP21ex);
	p->pP21ex = NULL;
	hipFree(p->pP21in);
	p->pP21in = NULL;
	hipFree(p->pP20);
	p->pP20 = NULL;

	gpuFree(p->_fire_count);

	return 0;
}

int cudaLogRateIAF(void *cpu, void *gpu, const char *name)
{
	IAFData *c = static_cast<IAFData *>(cpu);
	IAFData *g = static_cast<IAFData *>(gpu);

	IAFData *t = FROMGPU(g, 1);
	COPYFROMGPU(c->_fire_count, t->_fire_count, c->num);
	return logRateIAF(cpu, name);
}

real * cudaGetVIAF(void *data) {
	IAFData *c_g_lif = FROMGPU(static_cast<IAFData *>(data), 1);
	return c_g_lif->pV_m;
}
