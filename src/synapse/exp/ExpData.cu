#include <stdlib.h>
#include <string.h>
#include "../../../msg_utils/helper/helper_c.h"
#include "../../../msg_utils/helper/helper_gpu.h"
#include "ExpData.h"

void *cudaMallocExp()
{
	void *ret = NULL;
	checkCudaErrors(hipMalloc((void**)&(ret), sizeof(ExpData)*1));
	checkCudaErrors(hipMemset(ret, 0, sizeof(ExpData)*1));
	return ret;
}

void *cudaAllocExp(void *pCPU, size_t num)
{
	void *ret = cudaMallocExp();
	void *tmp = cudaAllocExpPara(pCPU, num);
	checkCudaErrors(hipMemcpy(ret, tmp, sizeof(ExpData)*1, hipMemcpyHostToDevice));
	tmp = free_c(tmp);
	return ret;
}

void *cudaAllocExpPara(void *pCPU, size_t num)
{
	ExpData *p = (ExpData*)pCPU;
	ExpData *ret = (ExpData*)malloc(sizeof(ExpData)*1);
	memset(ret, 0, sizeof(ExpData)*1);

    checkCudaErrors(hipMalloc((void**)&(ret->s), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->s, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->s, p->s, sizeof(real)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(ret->weight), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->weight, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->weight, p->weight, sizeof(real)*num, hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void**)&(ret->g), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->g, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->g, p->g, sizeof(real)*num, hipMemcpyHostToDevice));

	return ret;
}

int cudaFetchExp(void *pCPU, void *pGPU, size_t num)
{
	ExpData *pTmp = (ExpData*)malloc(sizeof(ExpData)*1);
	memset(pTmp, 0, sizeof(ExpData)*1);
	checkCudaErrors(hipMemcpy(pTmp, pGPU, sizeof(ExpData)*1, hipMemcpyDeviceToHost));

	cudaExpParaFromGPU(pCPU, pTmp, num);
	return 0;
}

int cudaExpParaToGPU(void *pCPU, void *pGPU, size_t num)
{
	ExpData *pC = (ExpData*)pCPU;
	ExpData *pG = (ExpData*)pGPU;

    checkCudaErrors(hipMemcpy(pG->s, pC->s, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->weight, pC->weight, sizeof(real)*num, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(pG->g, pC->g, sizeof(real)*num, hipMemcpyHostToDevice));

	return 0;
}

int cudaExpParaFromGPU(void *pCPU, void *pGPU, size_t num)
{
	ExpData *pC = (ExpData*)pCPU;
	ExpData *pG = (ExpData*)pGPU;

    checkCudaErrors(hipMemcpy(pC->s, pG->s, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->weight, pG->weight, sizeof(real)*num, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(pC->g, pG->g, sizeof(real)*num, hipMemcpyDeviceToHost));

	return 0;
}

int cudaFreeExp(void *pGPU)
{
	ExpData *tmp = (ExpData*)malloc(sizeof(ExpData)*1);
	memset(tmp, 0, sizeof(ExpData)*1);
	checkCudaErrors(hipMemcpy(tmp, pGPU, sizeof(ExpData)*1, hipMemcpyDeviceToHost));
	cudaFreeExpPara(tmp);
	tmp = free_c(tmp);
	hipFree(pGPU);
	pGPU = NULL;
	return 0;
}

int cudaFreeExpPara(void *pGPU)
{
	ExpData *p = (ExpData*)pGPU;

    hipFree(p->s);
    p->s = NULL;
	hipFree(p->weight);
	p->weight = NULL;
    hipFree(p->g);
    p->g = NULL;

	return 0;
}

