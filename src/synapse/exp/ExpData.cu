#include <stdlib.h>
#include <string.h>
#include "../../../msg_utils/helper/helper_c.h"
#include "../../../msg_utils/helper/helper_gpu.h"
#include "ExpData.h"

void *cudaMallocExp()
{
	void *ret = NULL;
	checkCudaErrors(hipMalloc((void**)&(ret), sizeof(ExpData)*1));
	checkCudaErrors(hipMemset(ret, 0, sizeof(ExpData)*1));
	return ret;
}

void *cudaAllocExp(void *pCPU, size_t num)
{
	void *ret = cudaMallocExp();
	void *tmp = cudaAllocExpPara(pCPU, num);
	checkCudaErrors(hipMemcpy(ret, tmp, sizeof(ExpData)*1, hipMemcpyHostToDevice));
	tmp = free_c(tmp);
	return ret;
}

void *cudaAllocExpPara(void *pCPU, size_t num)
{
	ExpData *p = (ExpData*)pCPU;
	ExpData *ret = (ExpData*)malloc(sizeof(ExpData)*1);
	memset(ret, 0, sizeof(ExpData)*1);

    checkCudaErrors(hipMalloc((void**)&(ret->pS), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pS, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pS, p->pS, sizeof(real)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(ret->pWeight), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pWeight, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pWeight, p->pWeight, sizeof(real)*num, hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void**)&(ret->pG), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pG, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pG, p->pG, sizeof(real)*num, hipMemcpyHostToDevice));

	return ret;
}

int cudaFetchExp(void *pCPU, void *pGPU, size_t num)
{
	ExpData *pTmp = (ExpData*)malloc(sizeof(ExpData)*1);
	memset(pTmp, 0, sizeof(ExpData)*1);
	checkCudaErrors(hipMemcpy(pTmp, pGPU, sizeof(ExpData)*1, hipMemcpyDeviceToHost));

	cudaExpParaFromGPU(pCPU, pTmp, num);
	return 0;
}

int cudaExpParaToGPU(void *pCPU, void *pGPU, size_t num)
{
	ExpData *pC = (ExpData*)pCPU;
	ExpData *pG = (ExpData*)pGPU;

    checkCudaErrors(hipMemcpy(pG->pS, pC->pS, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pWeight, pC->pWeight, sizeof(real)*num, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(pG->pG, pC->pG, sizeof(real)*num, hipMemcpyHostToDevice));

	return 0;
}

int cudaExpParaFromGPU(void *pCPU, void *pGPU, size_t num)
{
	ExpData *pC = (ExpData*)pCPU;
	ExpData *pG = (ExpData*)pGPU;

    checkCudaErrors(hipMemcpy(pC->pS, pG->pS, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pWeight, pG->pWeight, sizeof(real)*num, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(pC->pG, pG->pG, sizeof(real)*num, hipMemcpyDeviceToHost));

	return 0;
}

int cudaFreeExp(void *pGPU)
{
	ExpData *tmp = (ExpData*)malloc(sizeof(ExpData)*1);
	memset(tmp, 0, sizeof(ExpData)*1);
	checkCudaErrors(hipMemcpy(tmp, pGPU, sizeof(ExpData)*1, hipMemcpyDeviceToHost));
	cudaFreeExpPara(tmp);
	tmp = free_c(tmp);
	hipFree(pGPU);
	pGPU = NULL;
	return 0;
}

int cudaFreeExpPara(void *pGPU)
{
	ExpData *p = (ExpData*)pGPU;

    hipFree(p->pS);
    p->pS = NULL;
	hipFree(p->pWeight);
	p->pWeight = NULL;
    hipFree(p->pG);
    p->pG = NULL;

	return 0;
}

