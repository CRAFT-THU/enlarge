#include "hip/hip_runtime.h"

#include "../../gpu_utils/runtime.h"

#include "PoissonData.h"
#include <hiprand/hiprand_kernel.h>


__global__ void update_dense_poisson_hit(Connection *connection, PoissonData *data, real *buffer, const uinteger_t *firedTable, const uinteger_t *firedTableSizes, size_t firedTableCap, size_t num, size_t start_id, int time)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;  
	for (int idx = 0; idx < data->num; idx += blockDim.x * gridDim.x) {
		int sid = tid + idx;  // sid: synapse id
		if (sid < data->num) {
			hiprandState localState = data->pState[sid];
			int tmp = hiprand_poisson(&localState, data->pMean[connection->pSidMap[sid]]);
			data->pState[sid] = localState;
			atomicAdd(&(buffer[connection->dst[sid]]), tmp * data->pWeight[connection->pSidMap[sid]]);
		}
		// __syncthreads();
	}	
}

// __global__ void curand_setup_poisson_init_state(PoissonData *data)
// {
// 	int tid = threadIdx.x + blockIdx.x * blockDim.x;
// 	// if (id < num) {
// 	// 	hiprand_init(1234, id, 0, &state[id]);
// 	// }
// 	for (int idx = 0; idx < data->num; idx += blockDim.x * gridDim.x) {
// 		int sid = tid + idx;
// 		if (sid < data->num) {
// 			hiprand_init(666, sid, 0, &(data->pState[sid]));
// 		}
// 	}
// }

void cudaUpdatePoisson(Connection * connection, void *data, real *buffer, uinteger_t *firedTable, uinteger_t *firedTableSizes, size_t firedTableCap, size_t num, size_t start_id, int time, BlockSize *pSize)
{
	std::cout << "poisson cuda kernel: " << std::endl;
	// pSize->gridSize = 1;
	// pSize->blockSize = 128;
	std::cout << pSize->gridSize << " " << pSize->blockSize << std::endl;
	// std::cout << "buffer: ";
	// for (int i = 0; i < num; ++i)
	// 	std::cout << buffer[i] << " ";
	// std::cout << std::endl;
	// curand_setup_poisson_init_state<<<pSize->gridSize, pSize->blockSize>>>((PoissonData *)data);
	update_dense_poisson_hit<<<pSize->gridSize, pSize->blockSize>>>((Connection *)connection,  (PoissonData *)data, buffer, firedTable, firedTableSizes, firedTableCap, num, start_id, time);
}
