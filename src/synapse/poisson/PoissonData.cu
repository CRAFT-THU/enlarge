#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <string.h>
#include "../../../msg_utils/helper/helper_gpu.h"
#include "PoissonData.h"

void *cudaMallocPoisson()
{
	void *ret = NULL;
	checkCudaErrors(hipMalloc((void**)&(ret), sizeof(PoissonData)*1));
	checkCudaErrors(hipMemset(ret, 0, sizeof(PoissonData)*1));
	return ret;
}

__global__ void curand_setup_poisson_init_state(PoissonData *data)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	for (int idx = 0; idx < data->num; idx += blockDim.x * gridDim.x) {
		int sid = tid + idx;
		if (sid < data->num) {
			hiprand_init(666, sid, 0, &(data->pState[sid]));
		}
	}
}

void *cudaAllocPoisson(void *pCPU, size_t num)
{
	void *ret = cudaMallocPoisson();
	void *tmp = cudaAllocPoissonPara(pCPU, num);
	checkCudaErrors(hipMemcpy(ret, tmp, sizeof(PoissonData)*1, hipMemcpyHostToDevice));
	free(tmp);
	tmp = NULL;
	curand_setup_poisson_init_state<<<1, 128>>>((PoissonData *)ret);
	return ret;
}

void *cudaAllocPoissonPara(void *pCPU, size_t num)
{
	PoissonData *p = (PoissonData*)pCPU;
	PoissonData *ret = (PoissonData*)malloc(sizeof(PoissonData)*1);
	memset(ret, 0, sizeof(PoissonData)*1);
	memcpy(ret, p, sizeof(PoissonData)*1);
	
	// copy poisson synapse weight from CPU to GPU 
	checkCudaErrors(hipMalloc((void**)&(ret->pWeight), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pWeight, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pWeight, p->pWeight, sizeof(real)*num, hipMemcpyHostToDevice));
	
	// copy poisson mean from CPU to GPU 
	checkCudaErrors(hipMalloc((void**)&(ret->pMean), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pMean, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pMean, p->pMean, sizeof(real)*num, hipMemcpyHostToDevice));

	// allocate hiprandState in GPU
	checkCudaErrors(hipMalloc((void**)&(ret->pState), sizeof(hiprandState)*num));
	checkCudaErrors(hipMemset(ret->pState, 0, sizeof(hiprandState)*num));

	return (void *)ret;
}

int cudaFetchPoisson(void *pCPU, void *pGPU, size_t num)
{
	PoissonData *pTmp = (PoissonData*)malloc(sizeof(PoissonData)*1);
	memset(pTmp, 0, sizeof(PoissonData)*1);
	checkCudaErrors(hipMemcpy(pTmp, pGPU, sizeof(PoissonData)*1, hipMemcpyDeviceToHost));

	cudaPoissonParaFromGPU(pCPU, pTmp, num);
	return 0;
}

int cudaPoissonParaToGPU(void *pCPU, void *pGPU, size_t num)
{
	PoissonData *pC = (PoissonData*)pCPU;
	PoissonData *pG = (PoissonData*)pGPU;

	//checkCudaErrors(hipMemcpy(pG->pDst, pC->pDst, sizeof(int)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(pG->pWeight, pC->pWeight, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pMean, pC->pMean, sizeof(real)*num, hipMemcpyHostToDevice));
	// checkCudaErrors(hipMemcpy(pG->pState, pC->pState, sizeof(hiprandState)*num, hipMemcpyHostToDevice));

	return 0;
}

int cudaPoissonParaFromGPU(void *pCPU, void *pGPU, size_t num)
{
	PoissonData *pC = (PoissonData*)pCPU;
	PoissonData *pG = (PoissonData*)pGPU;

	// checkCudaErrors(hipMemcpy(pC->pDst, pG->pDst, sizeof(int)*num, hipMemcpyDeviceToHost));

	checkCudaErrors(hipMemcpy(pC->pWeight, pG->pWeight, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pMean, pG->pMean, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pState, pG->pState, sizeof(hiprandState)*num, hipMemcpyDeviceToHost));

	return 0;
}

int cudaFreePoisson(void *pGPU)
{
	PoissonData *tmp = (PoissonData*)malloc(sizeof(PoissonData)*1);
	memset(tmp, 0, sizeof(PoissonData)*1);
	checkCudaErrors(hipMemcpy(tmp, pGPU, sizeof(PoissonData)*1, hipMemcpyDeviceToHost));
	cudaFreePoissonPara(tmp);
	free(tmp);
	tmp = NULL;
	hipFree(pGPU);
	pGPU = NULL;
	return 0;
}

int cudaFreePoissonPara(void *pGPU)
{
	PoissonData *p = (PoissonData*)pGPU;
	// hipFree(p->pDst);
	// p->pDst = NULL;

	hipFree(p->pWeight);
	hipFree(p->pMean);
	hipFree(p->pState);
	p->pWeight = NULL;
	p->pMean = NULL;
	p->pState = NULL;

	return 0;
}

