#include <stdlib.h>
#include <string.h>
#include "../../../msg_utils/helper/helper_gpu.h"
#include "NMDASynData.h"
#include <helper_c.h>

void *cudaMallocNMDASyn()
{
	void *ret = NULL;
	checkCudaErrors(hipMalloc((void**)&(ret), sizeof(NMDASynData)*1));
	checkCudaErrors(hipMemset(ret, 0, sizeof(NMDASynData)*1));
	return ret;
}

void *cudaAllocNMDASyn(void *pCPU, size_t num)
{
	void *ret = cudaMallocNMDASyn();
	void *tmp = cudaAllocNMDASynPara(pCPU, num);
	checkCudaErrors(hipMemcpy(ret, tmp, sizeof(NMDASynData)*1, hipMemcpyHostToDevice));
	free(tmp);
	tmp = NULL;
	return ret;
}

void *cudaAllocNMDASynPara(void *pCPU, size_t num)
{
	NMDASynData *p = (NMDASynData*)pCPU;
	NMDASynData *ret = (NMDASynData*)malloc(sizeof(NMDASynData)*1);
	memset(ret, 0, sizeof(NMDASynData)*1);

	checkCudaErrors(hipMalloc((void**)&(ret->g), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->g, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->g, p->g, sizeof(real)*num, hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void**)&(ret->M_ca_coeff), sizeof(real)*num));
    checkCudaErrors(hipMemset(ret->M_ca_coeff, 0, sizeof(real)*num));
    checkCudaErrors(hipMemcpy(ret->M_ca_coeff, p->M_ca_coeff, sizeof(real)*num, hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void**)&(ret->M_c), sizeof(real)*num));
    checkCudaErrors(hipMemset(ret->M_c, 0, sizeof(real)*num));
    checkCudaErrors(hipMemcpy(ret->M_c, p->M_c, sizeof(real)*num, hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void**)&(ret->E_syn), sizeof(real)*num));
    checkCudaErrors(hipMemset(ret->E_syn, 0, sizeof(real)*num));
    checkCudaErrors(hipMemcpy(ret->E_syn, p->E_syn, sizeof(real)*num, hipMemcpyHostToDevice));

	return ret;
}

int cudaFetchNMDASyn(void *pCPU, void *pGPU, size_t num)
{
	NMDASynData *pTmp = (NMDASynData*)malloc(sizeof(NMDASynData)*1);
	memset(pTmp, 0, sizeof(NMDASynData)*1);
	checkCudaErrors(hipMemcpy(pTmp, pGPU, sizeof(NMDASynData)*1, hipMemcpyDeviceToHost));

	cudaNMDASynParaFromGPU(pCPU, pTmp, num);
	return 0;
}

int cudaNMDASynParaToGPU(void *pCPU, void *pGPU, size_t num)
{
	NMDASynData *pC = (NMDASynData*)pCPU;
	NMDASynData *pG = (NMDASynData*)pGPU;

	checkCudaErrors(hipMemcpy(pG->g, pC->g, sizeof(real)*num, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(pG->M_ca_coeff, pC->M_ca_coeff, sizeof(real)*num, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(pG->M_c, pC->M_c, sizeof(real)*num, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(pG->E_syn, pC->E_syn, sizeof(real)*num, hipMemcpyHostToDevice));

	return 0;
}

int cudaNMDASynParaFromGPU(void *pCPU, void *pGPU, size_t num)
{
	NMDASynData *pC = (NMDASynData*)pCPU;
	NMDASynData *pG = (NMDASynData*)pGPU;

	checkCudaErrors(hipMemcpy(pC->g, pG->g, sizeof(real)*num, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(pC->M_ca_coeff, pG->M_ca_coeff, sizeof(real)*num, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(pC->M_c, pG->M_c, sizeof(real)*num, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(pC->E_syn, pG->E_syn, sizeof(real)*num, hipMemcpyDeviceToHost));

	return 0;
}

int cudaFreeNMDASyn(void *pGPU)
{
	NMDASynData *tmp = (NMDASynData*)malloc(sizeof(NMDASynData)*1);
	memset(tmp, 0, sizeof(NMDASynData)*1);
	checkCudaErrors(hipMemcpy(tmp, pGPU, sizeof(NMDASynData)*1, hipMemcpyDeviceToHost));
	cudaFreeNMDASynPara(tmp);

	tmp = free_c(tmp);
	hipFree(pGPU); pGPU = NULL;

	return 0;
}

int cudaFreeNMDASynPara(void *pGPU)
{
	NMDASynData *p = (NMDASynData*)pGPU;

	hipFree(p->g); p->g = NULL;
    hipFree(p->M_ca_coeff); p->M_ca_coeff = NULL;
    hipFree(p->M_c); p->M_c = NULL;
    hipFree(p->E_syn); p->E_syn = NULL;

	return 0;
}

