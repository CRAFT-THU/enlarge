#include <stdlib.h>
#include <string.h>
#include "../../../msg_utils/helper/helper_c.h"
#include "../../../msg_utils/helper/helper_gpu.h"
#include "NMDAData.h"

void *cudaMallocNMDA()
{
	void *ret = NULL;
	checkCudaErrors(hipMalloc((void**)&(ret), sizeof(NMDAData)*1));
	checkCudaErrors(hipMemset(ret, 0, sizeof(NMDAData)*1));
	return ret;
}

void *cudaAllocNMDA(void *pCPU, size_t num)
{
	void *ret = cudaMallocNMDA();
	void *tmp = cudaAllocNMDAPara(pCPU, num);
	checkCudaErrors(hipMemcpy(ret, tmp, sizeof(NMDAData)*1, hipMemcpyHostToDevice));
	tmp = free_c(tmp);
	return ret;
}

void *cudaAllocNMDAPara(void *pCPU, size_t num)
{
	NMDAData *p = (NMDAData*)pCPU;
	NMDAData *ret = (NMDAData*)malloc(sizeof(NMDAData)*1);
	memset(ret, 0, sizeof(NMDAData)*1);

    checkCudaErrors(hipMalloc((void**)&(ret->pS), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pS, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pS, p->pS, sizeof(real)*num, hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void**)&(ret->pX), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pX, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pX, p->pX, sizeof(real)*num, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(ret->pC_decay), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pC_decay, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pC_decay, p->pC_decay, sizeof(real)*num, hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void**)&(ret->pC_rise), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pC_rise, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pC_rise, p->pC_rise, sizeof(real)*num, hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void**)&(ret->pG), sizeof(real)*num));
	checkCudaErrors(hipMemset(ret->pG, 0, sizeof(real)*num));
	checkCudaErrors(hipMemcpy(ret->pG, p->pG, sizeof(real)*num, hipMemcpyHostToDevice));

	return ret;
}

int cudaFetchNMDA(void *pCPU, void *pGPU, size_t num)
{
	NMDAData *pTmp = (NMDAData*)malloc(sizeof(NMDAData)*1);
	memset(pTmp, 0, sizeof(NMDAData)*1);
	checkCudaErrors(hipMemcpy(pTmp, pGPU, sizeof(NMDAData)*1, hipMemcpyDeviceToHost));

	cudaNMDAParaFromGPU(pCPU, pTmp, num);
	return 0;
}

int cudaNMDAParaToGPU(void *pCPU, void *pGPU, size_t num)
{
	NMDAData *pC = (NMDAData*)pCPU;
	NMDAData *pG = (NMDAData*)pGPU;

    checkCudaErrors(hipMemcpy(pG->pS, pC->pS, sizeof(real)*num, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pG->pX, pC->pX, sizeof(real)*num, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(pG->pC_decay, pC->pC_decay, sizeof(real)*num, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(pG->pC_rise, pC->pC_rise, sizeof(real)*num, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(pG->pG, pC->pG, sizeof(real)*num, hipMemcpyHostToDevice));

	return 0;
}

int cudaNMDAParaFromGPU(void *pCPU, void *pGPU, size_t num)
{
	NMDAData *pC = (NMDAData*)pCPU;
	NMDAData *pG = (NMDAData*)pGPU;

    checkCudaErrors(hipMemcpy(pC->pS, pG->pS, sizeof(real)*num, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(pC->pX, pG->pX, sizeof(real)*num, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pC->pC_decay, pG->pC_decay, sizeof(real)*num, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(pC->pC_rise, pG->pC_rise, sizeof(real)*num, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(pC->pG, pG->pG, sizeof(real)*num, hipMemcpyDeviceToHost));

	return 0;
}

int cudaFreeNMDA(void *pGPU)
{
	NMDAData *tmp = (NMDAData*)malloc(sizeof(NMDAData)*1);
	memset(tmp, 0, sizeof(NMDAData)*1);
	checkCudaErrors(hipMemcpy(tmp, pGPU, sizeof(NMDAData)*1, hipMemcpyDeviceToHost));
	cudaFreeNMDAPara(tmp);
	tmp = free_c(tmp);
	hipFree(pGPU);
	pGPU = NULL;
	return 0;
}

int cudaFreeNMDAPara(void *pGPU)
{
	NMDAData *p = (NMDAData*)pGPU;

    hipFree(p->pS);
    p->pS = NULL;
    hipFree(p->pX);
    p->pX = NULL;
	hipFree(p->pC_decay);
	p->pC_decay = NULL;
    hipFree(p->pC_rise);
    p->pC_rise = NULL;
    hipFree(p->pG);
    p->pG = NULL;

	return 0;
}

