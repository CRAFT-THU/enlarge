
#include <stdlib.h>
#include <assert.h>

#include "../gpu_utils/mem_op.h"
#include "Connection.h"

Connection * cudaAllocConnection(Connection * pCPU)
{
	size_t nNum = pCPU->nNum;
	size_t sNum = pCPU->sNum;
	size_t length = (pCPU->maxDelay - pCPU->minDelay + 1) * nNum;

	Connection * pGPU = NULL;
	Connection *pTmp = (Connection*)malloc(sizeof(Connection));
	pTmp->nNum = nNum;
	pTmp->sNum = sNum;
	pTmp->maxDelay = pCPU->maxDelay;
	pTmp->minDelay = pCPU->minDelay;

	checkCudaErrors(hipMalloc((void**)&(pTmp->pDelayStart), sizeof(size_t)*length));
	checkCudaErrors(hipMemcpy(pTmp->pDelayStart, pCPU->pDelayStart, sizeof(size_t)*length, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pTmp->pDelayNum), sizeof(size_t)*length));
	checkCudaErrors(hipMemcpy(pTmp->pDelayNum, pCPU->pDelayNum, sizeof(size_t)*length, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pTmp->pSidMap), sizeof(size_t)*sNum));
	checkCudaErrors(hipMemcpy(pTmp->pSidMap, pCPU->pSidMap, sizeof(size_t)*sNum, hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&(pTmp->dst), sizeof(size_t)*sNum));
	checkCudaErrors(hipMemcpy(pTmp->dst, pCPU->dst, sizeof(size_t)*sNum, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pTmp->pDelayStartRev), sizeof(size_t)*length));
	checkCudaErrors(hipMemcpy(pTmp->pDelayStartRev, pCPU->pDelayStartRev, sizeof(size_t)*length, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pTmp->pDelayNumRev), sizeof(size_t)*length));
	checkCudaErrors(hipMemcpy(pTmp->pDelayNumRev, pCPU->pDelayNumRev, sizeof(size_t)*length, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pTmp->pSidMapRev), sizeof(size_t)*sNum));
	checkCudaErrors(hipMemcpy(pTmp->pSidMapRev, pCPU->pSidMapRev, sizeof(size_t)*sNum, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc((void**)&(pGPU), sizeof(Connection)));
	checkCudaErrors(hipMemcpy(pGPU, pTmp, sizeof(Connection), hipMemcpyHostToDevice));

	free(pTmp);
	pTmp = NULL;

	return pGPU;
}

int cudaFetchConnection(Connection *pCPU, Connection *pGPU)
{
	size_t nNum = pCPU->nNum;
	size_t sNum = pCPU->sNum;
	size_t length = (pCPU->maxDelay - pCPU->minDelay + 1) * nNum;

	Connection *pTmp = (Connection*)malloc(sizeof(Connection));
	checkCudaErrors(hipMemcpy(pTmp, pGPU, sizeof(Connection), hipMemcpyDeviceToHost));
	assert(nNum == pTmp->nNum);
	assert(sNum == pTmp->sNum);
	assert(pCPU->maxDelay == pTmp->maxDelay);
	assert(pCPU->minDelay == pTmp->minDelay);

	checkCudaErrors(hipMemcpy(pCPU->pDelayStart, pTmp->pDelayStart, sizeof(size_t)*length, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pCPU->pDelayNum, pTmp->pDelayNum, sizeof(size_t)*length, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pCPU->pSidMap, pTmp->pSidMap, sizeof(size_t)*sNum, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pCPU->dst, pTmp->dst, sizeof(size_t)*sNum, hipMemcpyDeviceToHost));


	checkCudaErrors(hipMemcpy(pCPU->pDelayStartRev, pTmp->pDelayStartRev, sizeof(size_t)*length, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pCPU->pDelayNumRev, pTmp->pDelayNumRev, sizeof(size_t)*length, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pCPU->pSidMapRev, pTmp->pSidMapRev, sizeof(size_t)*sNum, hipMemcpyDeviceToHost));

	free(pTmp);
	pTmp = NULL;

	return 0;
}

int cudaFreeConnection(Connection *pGPU)
{
	Connection * pTmp = (Connection*)malloc(sizeof(Connection));
	checkCudaErrors(hipMemcpy(pTmp, pGPU, sizeof(Connection), hipMemcpyDeviceToHost));
	checkCudaErrors(hipFree(pTmp->pDelayStart));
	checkCudaErrors(hipFree(pTmp->pDelayNum));
	checkCudaErrors(hipFree(pTmp->pSidMap));
	checkCudaErrors(hipFree(pTmp->dst));
	checkCudaErrors(hipFree(pTmp->pDelayStartRev));
	checkCudaErrors(hipFree(pTmp->pDelayNumRev));
	checkCudaErrors(hipFree(pTmp->pSidMapRev));
	free(pTmp);
	pTmp = NULL;
	return 0;
}
