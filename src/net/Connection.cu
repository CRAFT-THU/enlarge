
#include <stdlib.h>
#include <assert.h>

#include "../../msg_utils/helper/helper_c.h"
#include "../../msg_utils/helper/helper_gpu.h"
#include "Connection.h"

Connection * cudaAllocConnection(Connection * pCPU)
{
	size_t nNum = pCPU->nNum;
	size_t sNum = pCPU->sNum;
	size_t length = (pCPU->maxDelay - pCPU->minDelay + 1) * nNum;

	Connection * pGPU = NULL;
	Connection *pTmp = malloc_c<Connection>();
	pTmp->nNum = nNum;
	pTmp->sNum = sNum;
	pTmp->maxDelay = pCPU->maxDelay;
	pTmp->minDelay = pCPU->minDelay;

	pTmp->pDelayStart = TOGPU(pCPU->pDelayStart, length+1);
	pTmp->pDelayNum = TOGPU(pCPU->pDelayNum, length);
	pTmp->pSidMap = TOGPU(pCPU->pSidMap, sNum);
	pTmp->dst = TOGPU(pCPU->dst, sNum);

	// pTmp->pDelayStartRev = copyToGPU(pCPU->pDelayStartRev, length+1);

	// checkCudaErrors(hipMalloc((void**)&(pTmp->pDelayNumRev), sizeof(size_t)*length));
	// checkCudaErrors(hipMemcpy(pTmp->pDelayNumRev, pCPU->pDelayNumRev, sizeof(size_t)*length, hipMemcpyHostToDevice));

	// checkCudaErrors(hipMalloc((void**)&(pTmp->pSidMapRev), sizeof(size_t)*sNum));
	// checkCudaErrors(hipMemcpy(pTmp->pSidMapRev, pCPU->pSidMapRev, sizeof(size_t)*sNum, hipMemcpyHostToDevice));

	pGPU = TOGPU(pTmp, 1);

	free(pTmp);
	pTmp = NULL;

	return pGPU;
}

int cudaFetchConnection(Connection *pCPU, Connection *pGPU)
{
	size_t nNum = pCPU->nNum;
	size_t sNum = pCPU->sNum;
	size_t length = (pCPU->maxDelay - pCPU->minDelay + 1) * nNum;

	Connection *pTmp = malloc_c<Connection>();
	COPYFROMGPU(pTmp, pGPU, 1);
	// checkCudaErrors(hipMemcpy(pTmp, pGPU, sizeof(Connection), hipMemcpyDeviceToHost));
	assert(nNum == pTmp->nNum);
	assert(sNum == pTmp->sNum);
	assert(pCPU->maxDelay == pTmp->maxDelay);
	assert(pCPU->minDelay == pTmp->minDelay);

	COPYFROMGPU(pCPU->pDelayStart, pTmp->pDelayStart, length+1);
	COPYFROMGPU(pCPU->pDelayNum, pTmp->pDelayNum, length);
	COPYFROMGPU(pCPU->pSidMap, pTmp->pSidMap, sNum);
	COPYFROMGPU(pCPU->dst, pTmp->dst, sNum);
	// checkCudaErrors(hipMemcpy(pCPU->pDelayStart, pTmp->pDelayStart, sizeof(size_t)*(length+1), hipMemcpyDeviceToHost));
	// checkCudaErrors(hipMemcpy(pCPU->pDelayNum, pTmp->pDelayNum, sizeof(size_t)*length, hipMemcpyDeviceToHost));
	// checkCudaErrors(hipMemcpy(pCPU->pSidMap, pTmp->pSidMap, sizeof(size_t)*sNum, hipMemcpyDeviceToHost));
	// checkCudaErrors(hipMemcpy(pCPU->dst, pTmp->dst, sizeof(size_t)*sNum, hipMemcpyDeviceToHost));


	// checkCudaErrors(hipMemcpy(pCPU->pDelayStartRev, pTmp->pDelayStartRev, sizeof(size_t)*(length+1), hipMemcpyDeviceToHost));
	// checkCudaErrors(hipMemcpy(pCPU->pDelayNumRev, pTmp->pDelayNumRev, sizeof(size_t)*length, hipMemcpyDeviceToHost));
	// checkCudaErrors(hipMemcpy(pCPU->pSidMapRev, pTmp->pSidMapRev, sizeof(size_t)*sNum, hipMemcpyDeviceToHost));

	free(pTmp);
	pTmp = NULL;

	return 0;
}

int cudaFreeConnection(Connection *pGPU)
{
	Connection * pTmp = malloc_c<Connection>();

	COPYFROMGPU(pTmp, pGPU, 1);
	// checkCudaErrors(hipMemcpy(pTmp, pGPU, sizeof(Connection), hipMemcpyDeviceToHost));
	gpuFree(pTmp->pDelayStart);
	gpuFree(pTmp->pDelayNum);
	gpuFree(pTmp->pSidMap);
	gpuFree(pTmp->dst);
	// checkCudaErrors(hipFree(pTmp->pDelayStartRev));
	// checkCudaErrors(hipFree(pTmp->pDelayNumRev));
	// checkCudaErrors(hipFree(pTmp->pSidMapRev));
	free(pTmp);
	pTmp = NULL;
	return 0;
}
